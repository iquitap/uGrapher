#include "gpu_intrinsics.h"
#include <hip/hip_cooperative_groups.h>

// total feature size
int32_t sz_fhid;		// (model_fhid * model_vertex)

int32_t sz_w2;          // (model_fhid * model_fhid)

#define eps                 0.0

using namespace cooperative_groups;
int32_t __delta_param;
gpu_runtime::GraphT __device__ edges;
gpu_runtime::GraphT __host_edges;
gpu_runtime::GraphT __device__ edges__transposed;
gpu_runtime::GraphT __host_edges__transposed;

int32_t __device__ M; 
int32_t __host_M;

int32_t __device__ input_f_size; 
int32_t __host_input_f_size;

int32_t __device__ hidden_f_size; 
int32_t __host_hidden_f_size;

int32_t __device__ output_f_size; 
int32_t __host_output_f_size;

int32_t __device__ vector_size; 
int32_t __host_vector_size;

float __device__ *feature_rst_1;
float *__host_feature_rst_1;
float *__device_feature_rst_1;

float __device__ *feature_input_out_2;
float *__host_feature_input_out_2;
float *__device_feature_input_out_2;

float __device__ *feature_input_eps_2;
float *__host_feature_input_eps_2;
float *__device_feature_input_eps_2;

float __device__ *feature_rst_2;
float *__host_feature_rst_2;
float *__device_feature_rst_2;

float __device__ *weight_2;
float *__host_weight_2;
float *__device_weight_2;

void __device__ init_feature(int32_t v, int32_t f);
void __device__ init_feature(int32_t v, int32_t f) {
	int32_t f_size = hidden_f_size;
	feature_rst_1[f] = ((v+1)%10 - f%f_size%10)/10000.0;
}

//layer-2
void __device__ gin_2(int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size);
//template <typename EdgeWeightType>
void __device__ gpu_operator_body_1(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {

    gin_2(dst, src, feat, A, C, Feat_Size);
}
void __device__ gin_2(int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size) {
    gpu_runtime::writeAdd(&C[((dst * Feat_Size) + feat)], A[((src * Feat_Size) + feat)]);
}

void __device__ gin_2_nAtm(int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size);
//template <typename EdgeWeightType>
void __device__ gpu_operator_body_1_nAtm(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
    
    gin_2_nAtm(dst, src, feat, A, C, Feat_Size);
}
void __device__ gin_2_nAtm(int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size) {
    C[((dst * Feat_Size) + feat)] += A[((src * Feat_Size) + feat)];
}

void __device__ eps_update_2(int32_t v, int32_t f);
void __device__ eps_update_2(int32_t v, int32_t f) {
    feature_input_eps_2[f] = (1 + eps) * feature_rst_1[f] + feature_input_out_2[f];
}

int __host__ main(int argc, char* argv[]) {
    hipError_t cuda_error;
    hipblasStatus_t cublas_error;
    // Initialize CUDA
    cuda_error = hipSetDevice(0);
    if (cuda_error != hipSuccess) {
        cout << "Error: hipSetDevice failed!" << endl;
        return EXIT_FAILURE;
    }

    // Initialize CUBLAS
    // hipblasHandle_t cublas_handle;

    const int GPUNUM = 1;
    hipblasHandle_t* cublasHs = new hipblasHandle_t[GPUNUM];
    for (int i = 0; i < GPUNUM; i++) {
        cublas_error = hipblasCreate(&cublasHs[i]);
        if (cublas_error != HIPBLAS_STATUS_SUCCESS) {
            cout << "Error: hipblasCreate failed!" << endl;
            return EXIT_FAILURE;
        }
    }

	__delta_param = 1;
	gpu_runtime::load_graph(__host_edges, argv[1], false);
	hipMemcpyToSymbol(HIP_SYMBOL(edges), &__host_edges, sizeof(__host_edges), 0, hipMemcpyHostToDevice);

    __host_edges__transposed = gpu_runtime::builtin_transpose(__host_edges);
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(edges__transposed), &__host_edges__transposed, sizeof(__host_edges__transposed), 0, hipMemcpyHostToDevice));

	__host_M = 128;
	hipMemcpyToSymbol(HIP_SYMBOL(M), &__host_M, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_input_f_size = std::stoi(argv[2]);
	hipMemcpyToSymbol(HIP_SYMBOL(input_f_size), &__host_input_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_hidden_f_size = std::stoi(argv[3]);
	hipMemcpyToSymbol(HIP_SYMBOL(hidden_f_size), &__host_hidden_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_output_f_size = std::stoi(argv[4]);
	hipMemcpyToSymbol(HIP_SYMBOL(output_f_size), &__host_output_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_vector_size = __host_edges__transposed.num_vertices;
    hipMemcpyToSymbol(HIP_SYMBOL(vector_size), &__host_vector_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	
    sz_fhid = __host_hidden_f_size * __host_vector_size;

    sz_w2 = __host_hidden_f_size * __host_hidden_f_size;
    
    //layer-1
	hipMalloc(&__device_feature_rst_1, sz_fhid * sizeof(float));
    hipMemset(__device_feature_rst_1, 0, sz_fhid * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(feature_rst_1), &__device_feature_rst_1, sizeof(float*), 0);
	__host_feature_rst_1 = new float[sz_fhid];

    //layer-2
	hipMalloc(&__device_feature_input_out_2, sz_fhid * sizeof(float));
    hipMemset(__device_feature_input_out_2, 0, sz_fhid * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(feature_input_out_2), &__device_feature_input_out_2, sizeof(float*), 0);
	__host_feature_input_out_2 = new float[sz_fhid];

	hipMalloc(&__device_feature_input_eps_2, sz_fhid * sizeof(float));
    hipMemset(__device_feature_input_eps_2, 0, sz_fhid * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(feature_input_eps_2), &__device_feature_input_eps_2, sizeof(float*), 0);
	__host_feature_input_eps_2 = new float[sz_fhid];

	hipMalloc(&__device_feature_rst_2, sz_fhid * sizeof(float));
    hipMemset(__device_feature_rst_2, 0, sz_fhid * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(feature_rst_2), &__device_feature_rst_2, sizeof(float*), 0);
	__host_feature_rst_2 = new float[sz_fhid];

	hipMalloc(&__device_weight_2, sz_w2 * sizeof(float));
    hipMemset(__device_weight_2, 0, sz_w2 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(weight_2), &__device_weight_2, sizeof(float*), 0);
	__host_weight_2 = new float[sz_w2];

    // printf("init __host_weight_2:%d %d %d\n", __host_hidden_f_size, __host_hidden_f_size, __host_hidden_f_size*__host_hidden_f_size);
    for (int32_t i = 0; i < __host_hidden_f_size; i++) {
        for (int32_t j = 0; j < __host_hidden_f_size; j++) {
            __host_weight_2[((i * __host_hidden_f_size) + j)] = (i%10 - j%10)/10000.0;
            CUDA_CALL(hipMemcpy(__device_weight_2 + ((i * __host_hidden_f_size) + j), __host_weight_2 + ((i * __host_hidden_f_size) + j), sizeof(float), hipMemcpyHostToDevice));
        }
    }

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(cuda_error));
    }

    bool run_vf = std::stoi(argv[5]);
    bool run_ef = std::stoi(argv[6]);

    // decide load balance schedule
	gpu_runtime::gnn_load_balance_type lb_c;
    int32_t ef_num_cta=0;
    int32_t ef_feat_size=__host_hidden_f_size;
    int32_t group_size=std::stoi(argv[9]);
    int32_t par_tiling=std::stoi(argv[10]);
    gpu_runtime::uGrapher_init<gpu_operator_body_1, gpu_operator_body_1_nAtm>(
        0, argv[8], &lb_c, __host_edges__transposed, &ef_num_cta, 
        &ef_feat_size, &group_size, &par_tiling);
    
    gpu_runtime::f_vertex_set_apply_host<init_feature>(__host_edges__transposed, __host_hidden_f_size);
    
    int32_t vf_num_cta;
	int32_t vf_feat_size;
    vf_feat_size = __host_hidden_f_size;
    gpu_runtime::f_vertex_set_apply_info(__host_edges__transposed, vf_num_cta, vf_feat_size);
    
    float time = 0, time_vf = 0, time_ef = 0;
    float elapsed_time;
    int cnt = 20;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	for (int32_t trail = 0; trail < cnt; trail++) {
        //layer-2
        if (run_ef) hipEventRecord(start);
        {
            gpu_runtime::uGrapher_exec(0, &lb_c, &ef_num_cta, 
                __host_edges__transposed, __device_feature_rst_1, 0, __device_feature_input_out_2, 
                &ef_feat_size, &group_size, &par_tiling);
            // hipDeviceSynchronize();
        }
        if (run_ef) {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed_time, start, stop);
		    time_ef = time_ef + elapsed_time;
        }

        if (run_vf) hipEventRecord(start);
        {
            gpu_runtime::f_vertex_set_apply_kernel<eps_update_2><<<vf_num_cta, CTA_SIZE>>>(__host_edges__transposed, vf_feat_size);
            // hipDeviceSynchronize();
        }
        if (run_vf) {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed_time, start, stop);
		    time_vf = time_vf + elapsed_time;
        }

        gpu_runtime::matmul_NN(cublasHs, __device_feature_input_eps_2, __device_weight_2, __device_feature_rst_2, __host_vector_size, __host_hidden_f_size, __host_hidden_f_size);
        hipDeviceSynchronize(); 
    }

    time_vf = time_vf * 1000 / cnt;
    time_ef = time_ef * 1000 / cnt;
    time = time_vf * run_vf + time_ef * run_ef;
    cout << "time: " << time << " msec" << endl;

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(cuda_error));
    }

    std::fstream fp;
    if (run_vf) {
        fp.open(argv[11], std::ios::out|std::ios::app);
        fp << time_vf;
        fp.close();
    }
    if (run_ef) {
        fp.open(argv[11], std::ios::out|std::ios::app);
        fp << time_ef;
        fp.close();
    }

    hipFree(__device_feature_input_out_2);
    hipFree(__device_feature_input_eps_2);
    hipFree(__device_weight_2);
    hipFree(__device_feature_rst_1);
    hipFree(__device_feature_rst_2);

    delete __host_feature_input_out_2;
    delete __host_feature_input_eps_2;
    delete __host_weight_2;
    delete __host_feature_rst_1;
    delete __host_feature_rst_2;
    
    hipEventDestroy(start);
    hipEventDestroy(stop);

    for (int i = 0; i < GPUNUM; i++) {
		hipblasDestroy(cublasHs[i]);
	}

	delete cublasHs;

    return 0;
}