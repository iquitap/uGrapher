#include "hip/hip_runtime.h"
#include "gpu_intrinsics.h"
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

#define sz_head         8

// total feature size
int32_t sz_fhid;		// (model_fhid * model_vertex)
int32_t sz_fout;		// (model_fout * model_vertex)

// weight size
int32_t sz_w2;       // (model_fhid * model_fout * sz_head)

int32_t sz_atten_2;  // (model_fout * sz_head)

// edge feature size
int32_t sz_e_2;              // (model_vertex)

int32_t sz_edge_feat_2;      // (model_edge)

// leaky_relu_param
#define negative_slope  0.01

int32_t __delta_param;
gpu_runtime::GraphT __device__ edges;
gpu_runtime::GraphT __host_edges;
gpu_runtime::GraphT __device__ edges__transposed;
gpu_runtime::GraphT __host_edges__transposed;

int32_t __device__ input_f_size; 
int32_t __host_input_f_size;

int32_t __device__ hidden_f_size; 
int32_t __host_hidden_f_size;

int32_t __device__ output_f_size; 
int32_t __host_output_f_size;

int32_t __device__ vector_size; 
int32_t __host_vector_size;

int32_t __device__ edge_size;
int32_t __host_edge_size;

int32_t __device__ num_head; 
int32_t __host_num_head;

float __device__ *rst_1;
float *__host_rst_1;
float *__device_rst_1;

float __device__ *Weight_2;
float *__host_Weight_2;
float *__device_Weight_2;

float __device__ *input_hidden_2;
float *__host_input_hidden_2;
float *__device_input_hidden_2;

float __device__ *Weight_src_2;
float *__host_Weight_src_2;
float *__device_Weight_src_2;

float __device__ *feature_el_2;
float *__host_feature_el_2;
float *__device_feature_el_2;

float __device__ *Weight_dst_2;
float *__host_Weight_dst_2;
float *__device_Weight_dst_2;

float __device__ *feature_er_2;
float *__host_feature_er_2;
float *__device_feature_er_2;

float __device__ *edge_exp_2;
float *__host_edge_exp_2;
float *__device_edge_exp_2;

float __device__ *edge_softmax_sum_2;
float *__host_edge_softmax_sum_2;
float *__device_edge_softmax_sum_2;

float __device__ *rst_2;
float *__host_rst_2;
float *__device_rst_2;

void __device__ init_feature(int32_t v, int32_t f);
void __device__ init_feature(int32_t v, int32_t f) {
	int32_t f_size = hidden_f_size * sz_head;
	rst_1[f] = (v+1 + f%f_size)/100000.0;
}

//layer-2

////////////////////////////////////////////////////////////////////////////////
// rewrite u_add_v_2
void __device__ u_add_v_2(int32_t src, int32_t dst, int32_t edge, float* A, float* B, float* C);
//template <typename EdgeWeightType>
void __device__ gpu_operator_body_30(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
    // Body of the actual operator
    u_add_v_2(dst, src, edge, A, B, C);
}
void __device__ u_add_v_2(int32_t src, int32_t dst, int32_t edge, float* A, float* B, float* C) {
    C[edge] = (A[src] + B[dst]);
}

void __device__ leak_relu_2(int32_t edge, float* A, float* C);
//template <typename EdgeWeightType>
void __device__ gpu_operator_body_31(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
    // Body of the actual operator
    leak_relu_2(edge, A, C);
}
void __device__ leak_relu_2(int32_t edge, float* A, float* C) {
    float tmp = A[edge];
    C[edge] = (tmp > 0.0) ? tmp : negative_slope * tmp;
}

void __device__ expf_2(int32_t edge, float* A, float* C);
//template <typename EdgeWeightType>
void __device__ gpu_operator_body_32(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
    // Body of the actual operator
    expf_2(edge, A, C);
}
void __device__ expf_2(int32_t edge, float* A, float* C) {
    float tmp = A[edge];
    C[edge] = __expf(tmp);
}
// rewrite u_add_v_e end
////////////////////////////////////////////////////////////////////////////////

void __device__ soft_max_rst_2(int32_t dst, int32_t edge, float* A, float* C);
//template <typename EdgeWeightType>
void __device__ gpu_operator_body_4(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
    // Body of the actual operator
    soft_max_rst_2(src, edge, A, C);
}
void __device__  soft_max_rst_2(int32_t dst, int32_t edge, float* A, float* C) {
    gpu_runtime::writeAdd(&C[dst], A[edge]);
}

////////////////////////////////////////////////////////////////////////////////
// rewrite get_rst_2
void __device__ fdiv2(int32_t dst, int32_t edge, float* A, float* B, float* C);
//template <typename EdgeWeightType>
void __device__ gpu_operator_body_50(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
    // Body of the actual operator
    fdiv2(src, edge, A, B, C);
}
void __device__ fdiv2(int32_t dst, int32_t edge, float* A, float* B, float* C) {
    float tmp = A[edge];
    C[edge] = __fdividef(tmp, B[dst]);
}

void __device__ get_rst_2(int32_t src, int32_t dst, int32_t edge, int32_t feat, float* A, float* B, float* C, int32_t Feat_Size);
//template <typename EdgeWeightType>
void __device__ gpu_operator_body_51(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
    // Body of the actual operator
    get_rst_2(dst, src, edge, feat, A, B, C, Feat_Size);
}
void __device__  get_rst_2(int32_t src, int32_t dst, int32_t edge, int32_t feat, float* A, float* B, float* C, int32_t Feat_Size) {
    float tmp = A[edge];
    gpu_runtime::writeAdd(&C[dst * Feat_Size + feat], tmp * B[src * Feat_Size + feat]);
}

void __device__ get_rst_2_nAtm(int32_t src, int32_t dst, int32_t edge, int32_t feat, float* A, float* B, float* C, int32_t Feat_Size);
//template <typename EdgeWeightType>
void __device__ gpu_operator_body_51_nAtm(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
    // Body of the actual operator
    get_rst_2_nAtm(dst, src, edge, feat, A, B, C, Feat_Size);
}
void __device__  get_rst_2_nAtm(int32_t src, int32_t dst, int32_t edge, int32_t feat, float* A, float* B, float* C, int32_t Feat_Size) {
    float tmp = A[edge];
    C[dst * Feat_Size + feat] += tmp * B[src * Feat_Size + feat];
}

////////////////////////////////////////////////////////////////////////////////

// ELU_2
void __device__ elu_2(int32_t v, int32_t f) {
    if(rst_2[f] < 0) {
        rst_2[f] = __expf(rst_2[f]) - 1;
    }
}

int __host__ main(int argc, char* argv[]) {
    hipError_t cuda_error;
    hipblasStatus_t cublas_error;
    // Initialize CUDA
    cuda_error = hipSetDevice(0);
    if (cuda_error != hipSuccess) {
        cout << "Error: hipSetDevice failed!" << endl;
        return EXIT_FAILURE;
    }

    // Initialize CUBLAS
    const int GPUNUM = 1;
    hipblasHandle_t* cublasHs = new hipblasHandle_t[GPUNUM];
    for (int i = 0; i < GPUNUM; i++) {
        cublas_error = hipblasCreate(&cublasHs[i]);
        if (cublas_error != HIPBLAS_STATUS_SUCCESS) {
            cout << "Error: hipblasCreate failed!" << endl;
            return EXIT_FAILURE;
        }
    }

	__delta_param = 1;
	gpu_runtime::load_graph(__host_edges, argv[1], false);
	hipMemcpyToSymbol(HIP_SYMBOL(edges), &__host_edges, sizeof(__host_edges), 0, hipMemcpyHostToDevice);

    // printf("Graph loaded\n");
    // printf("Graph: %d nodes, %d edges\n", __host_edges.num_vertices, __host_edges.num_edges);

	__host_edges__transposed = gpu_runtime::builtin_transpose(__host_edges);
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(edges__transposed), &__host_edges__transposed, sizeof(__host_edges__transposed), 0, hipMemcpyHostToDevice));

	__host_input_f_size = std::stoi(argv[2]);
	hipMemcpyToSymbol(HIP_SYMBOL(input_f_size), &__host_input_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_hidden_f_size = std::stoi(argv[3]);
	hipMemcpyToSymbol(HIP_SYMBOL(hidden_f_size), &__host_hidden_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_output_f_size = std::stoi(argv[4]);
	hipMemcpyToSymbol(HIP_SYMBOL(output_f_size), &__host_output_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_vector_size = __host_edges__transposed.num_vertices;
	hipMemcpyToSymbol(HIP_SYMBOL(vector_size), &__host_vector_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_edge_size = __host_edges__transposed.num_edges;
	hipMemcpyToSymbol(HIP_SYMBOL(edge_size), &__host_edge_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_num_head = sz_head;
	hipMemcpyToSymbol(HIP_SYMBOL(num_head), &__host_num_head, sizeof(int32_t), 0, hipMemcpyHostToDevice);

    sz_fhid = __host_hidden_f_size * __host_vector_size;
    sz_fout = __host_output_f_size * __host_vector_size;
	
    sz_w2 = __host_hidden_f_size * __host_output_f_size * sz_head;
	
    sz_atten_2 = __host_output_f_size * sz_head;
	
    sz_e_2 = __host_vector_size;

    sz_edge_feat_2 = __host_edge_size;

	hipMalloc(&__device_rst_1, sz_fhid * sz_head * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(rst_1), &__device_rst_1, sizeof(float*), 0);
	__host_rst_1 = new float[sz_fhid * sz_head];

	hipMalloc(&__device_Weight_2, sz_w2 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(Weight_2), &__device_Weight_2, sizeof(float*), 0);
	__host_Weight_2 = new float[sz_w2];

	hipMalloc(&__device_input_hidden_2, sz_fout * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(input_hidden_2), &__device_input_hidden_2, sizeof(float*), 0);
	__host_input_hidden_2 = new float[sz_fout];
	
    hipMalloc(&__device_Weight_src_2, sz_atten_2 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(Weight_src_2), &__device_Weight_src_2, sizeof(float*), 0);
	__host_Weight_src_2 = new float[sz_atten_2];
	
    hipMalloc(&__device_feature_el_2, sz_e_2 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(feature_el_2), &__device_feature_el_2, sizeof(float*), 0);
	__host_feature_el_2 = new float[sz_e_2];
	
    hipMalloc(&__device_Weight_dst_2, sz_atten_2 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(Weight_dst_2), &__device_Weight_dst_2, sizeof(float*), 0);
	__host_Weight_dst_2 = new float[sz_atten_2];

	hipMalloc(&__device_feature_er_2, sz_e_2 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(feature_er_2), &__device_feature_er_2, sizeof(float*), 0);
	__host_feature_er_2 = new float[sz_e_2];
	
    hipMalloc(&__device_edge_exp_2, sz_edge_feat_2 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(edge_exp_2), &__device_edge_exp_2, sizeof(float*), 0);
	__host_edge_exp_2 = new float[sz_edge_feat_2];

	hipMalloc(&__device_edge_softmax_sum_2, sz_e_2 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(edge_softmax_sum_2), &__device_edge_softmax_sum_2, sizeof(float*), 0);
	__host_edge_softmax_sum_2 = new float[sz_e_2];
	
    hipMalloc(&__device_rst_2, sz_fout * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(rst_2), &__device_rst_2, sizeof(float*), 0);
	__host_rst_2 = new float[sz_fout];

	gpu_runtime::f_vertex_set_apply_host<init_feature>(__host_edges__transposed, __host_hidden_f_size * sz_head);

    // printf("init __host_Weight_2:%d %d %d\n", __host_hidden_f_size, __host_output_f_size, __host_output_f_size*__host_hidden_f_size);
    initGPUData(__device_Weight_2, sz_w2, 0.004);
    // printf("init __host_Weight_src_2:%d %d %d\n", __host_output_f_size, __host_num_head, __host_num_head*__host_output_f_size);
    initGPUData(__device_Weight_src_2, sz_atten_2, 0.005);
    // printf("init __host_Weight_dst_2:%d %d %d\n", __host_output_f_size, __host_num_head, __host_num_head*__host_output_f_size);
    initGPUData(__device_Weight_dst_2, sz_atten_2, 0.006);

    bool run_vf = std::stoi(argv[5]);
    bool run_ef = std::stoi(argv[6]);

    // decide load balance schedule
	gpu_runtime::gnn_load_balance_type lb_c[6];
    int32_t ef_num_cta[6]={0,0,0,0,0,0};
    int32_t ef_feat_size[6]={1,1,1,1,1,__host_output_f_size};
    int32_t group_size[6]={1,1,1,1,1,std::stoi(argv[9])};
    int32_t par_tiling[6]={1,1,1,1,1,std::stoi(argv[10])};
    gpu_runtime::uGrapher_init<gpu_operator_body_30, gpu_operator_body_30>(
        0, "t_edge_group_tiling", lb_c, __host_edges__transposed, ef_num_cta, 
        ef_feat_size, group_size, par_tiling);
    gpu_runtime::uGrapher_init<gpu_operator_body_31, gpu_operator_body_31>(
        1, "t_edge_group_tiling", lb_c, __host_edges__transposed, ef_num_cta, 
        ef_feat_size, group_size, par_tiling);
    gpu_runtime::uGrapher_init<gpu_operator_body_32, gpu_operator_body_32>(
        2, "t_edge_group_tiling", lb_c, __host_edges__transposed, ef_num_cta, 
        ef_feat_size, group_size, par_tiling);
    gpu_runtime::uGrapher_init<gpu_operator_body_4, gpu_operator_body_4>(
        3, "t_edge_group_tiling", lb_c, __host_edges__transposed, ef_num_cta, 
        ef_feat_size, group_size, par_tiling);
    gpu_runtime::uGrapher_init<gpu_operator_body_50, gpu_operator_body_50>(
        4, "t_edge_group_tiling", lb_c, __host_edges__transposed, ef_num_cta, 
        ef_feat_size, group_size, par_tiling);
    gpu_runtime::uGrapher_init<gpu_operator_body_51, gpu_operator_body_51_nAtm>(
        5, argv[8], lb_c, __host_edges__transposed, ef_num_cta, 
        ef_feat_size, group_size, par_tiling);
    
    int32_t vf_num_cta;
    int32_t vf_feat_size;
    vf_feat_size = __host_output_f_size;
    gpu_runtime::f_vertex_set_apply_info(__host_edges__transposed, vf_num_cta, vf_feat_size);
        
    float time = 0, time_vf = 0, time_ef = 0;
    float elapsed_time;
    int cnt = 20;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	for (int32_t trail = 0; trail < cnt; trail++) {

        //layer_2
        // updateVertex_z_2
        gpu_runtime::matmul_NN(cublasHs, __device_rst_1, __device_Weight_2, __device_input_hidden_2, __host_vector_size, __host_output_f_size, __host_hidden_f_size * sz_head);

        //updateVertex_att_src_2
        gpu_runtime::matmul_NN(cublasHs, __device_input_hidden_2, __device_Weight_src_2, __device_feature_el_2, __host_vector_size, 1, __host_output_f_size);

        //updateVertex_att_dst_2
        gpu_runtime::matmul_NN(cublasHs, __device_input_hidden_2, __device_Weight_dst_2, __device_feature_er_2, __host_vector_size, 1, __host_output_f_size);
        hipDeviceSynchronize();

        {
            gpu_runtime::uGrapher_exec(0, lb_c, ef_num_cta, 
                __host_edges__transposed, __device_feature_el_2, __device_feature_er_2, __device_edge_exp_2, 
                ef_feat_size, group_size, par_tiling);
            hipDeviceSynchronize();
        }
        {
            gpu_runtime::uGrapher_exec(1, lb_c, ef_num_cta, 
                __host_edges__transposed, __device_edge_exp_2, 0, __device_edge_exp_2, 
                ef_feat_size, group_size, par_tiling);
            hipDeviceSynchronize();
        }
        {
            gpu_runtime::uGrapher_exec(2, lb_c, ef_num_cta, 
                __host_edges__transposed, __device_edge_exp_2, 0, __device_edge_exp_2, 
                ef_feat_size, group_size, par_tiling);
            hipDeviceSynchronize();
        }

        // exp
        {
            gpu_runtime::uGrapher_exec(3, lb_c, ef_num_cta, 
                __host_edges__transposed, __device_edge_exp_2, 0, __device_edge_softmax_sum_2, 
                ef_feat_size, group_size, par_tiling);
            hipDeviceSynchronize();
        }

        {
            gpu_runtime::uGrapher_exec(4, lb_c, ef_num_cta, 
                __host_edges__transposed, __device_edge_exp_2, __device_edge_softmax_sum_2, __device_edge_exp_2, 
                ef_feat_size, group_size, par_tiling);
            hipDeviceSynchronize();
        }

        if (run_ef) hipEventRecord(start);
        {
			gpu_runtime::uGrapher_exec(5, lb_c, ef_num_cta, 
                __host_edges__transposed, __device_edge_exp_2, __device_input_hidden_2, __device_rst_2, 
                ef_feat_size, group_size, par_tiling);
            // hipDeviceSynchronize();
		}
        if (run_ef) {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed_time, start, stop);
		    time_ef = time_ef + elapsed_time;
        }

        if (run_vf) hipEventRecord(start);
        {   
            gpu_runtime::f_vertex_set_apply_kernel<elu_2><<<vf_num_cta, CTA_SIZE>>>(__host_edges__transposed, vf_feat_size);
            // hipDeviceSynchronize();
	    }
        if (run_vf) {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed_time, start, stop);
		    time_vf = time_vf + elapsed_time;
        }
	}
    
    time_vf = time_vf * 1000 / cnt;
    time_ef = time_ef * 1000 / cnt;
    time = time_vf * run_vf + time_ef * run_ef;
    cout << "time: " << time << " msec" << endl;

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf(" 600 CUDA Error: %s\n", hipGetErrorString(cuda_error));
        return 0;
    }

    std::fstream fp;
    if (run_vf) {
        fp.open(argv[11], std::ios::out|std::ios::app);
        fp << time_vf;
        fp.close();
    }
    if (run_ef) {
        fp.open(argv[11], std::ios::out|std::ios::app);
        fp << time_ef;
        fp.close();
    }

    //free
    CUDA_CALL(hipFree(__device_rst_1));

    CUDA_CALL(hipFree(__device_Weight_2));
    CUDA_CALL(hipFree(__device_input_hidden_2));
    CUDA_CALL(hipFree(__device_Weight_src_2));
    CUDA_CALL(hipFree(__device_feature_el_2));
    CUDA_CALL(hipFree(__device_feature_er_2));
    CUDA_CALL(hipFree(__device_Weight_dst_2));
    CUDA_CALL(hipFree(__device_edge_exp_2));
    CUDA_CALL(hipFree(__device_edge_softmax_sum_2));
    CUDA_CALL(hipFree(__device_rst_2));

    delete __host_rst_1;

    delete __host_Weight_2;
    delete __host_input_hidden_2;
    delete __host_Weight_src_2;
    delete __host_feature_el_2;
    delete __host_feature_er_2;
    delete __host_Weight_dst_2;
    delete __host_edge_exp_2;
    delete __host_edge_softmax_sum_2;
    delete __host_rst_2;

    // cublas free
    for (int i = 0; i < GPUNUM; i++) {
		hipblasDestroy(cublasHs[i]);
	}

	delete cublasHs;
}