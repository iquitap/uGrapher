// full-batch
// cora
// aggregation op = sum/pool/lstm/mean
// this file is for mean aggregation
#include "gpu_intrinsics.h"
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

// total feature size
int32_t sz_fin;		    // (model_fin  * model_vertex)
int32_t sz_fhid;		// (model_fhid * model_vertex)

// weight size
int32_t sz_w1;          // (model_fin * model_fhid)

int32_t __delta_param;
gpu_runtime::GraphT __device__ edges;
gpu_runtime::GraphT __host_edges;
gpu_runtime::GraphT __device__ edges__transposed;
gpu_runtime::GraphT __host_edges__transposed;

int32_t __device__ M; 
int32_t __host_M;

int32_t __device__ input_f_size; 
int32_t __host_input_f_size;

int32_t __device__ hidden_f_size; 
int32_t __host_hidden_f_size;

int32_t __device__ output_f_size; 
int32_t __host_output_f_size;

int32_t __device__ vector_size; 
int32_t __host_vector_size;

float __device__ *feature_input;
float *__host_feature_input;
float *__device_feature_input;

float __device__ *feature_input_out_1;
float *__host_feature_input_out_1;
float *__device_feature_input_out_1;

float __device__ *feature_input_fc_1;
float *__host_feature_input_fc_1;
float *__device_feature_input_fc_1;

float __device__ *feature_input_fc_ngh_1;
float *__host_feature_input_fc_ngh_1;
float *__device_feature_input_fc_ngh_1;

float __device__ *feature_rst_1;
float *__host_feature_rst_1;
float *__device_feature_rst_1;

float __device__ *weight_fc_1;
float *__host_weight_fc_1;
float *__device_weight_fc_1;

float __device__ *weight_fc_ngh_1;
float *__host_weight_fc_ngh_1;
float *__device_weight_fc_ngh_1;

void __device__ sum_1(gpu_runtime::GraphT graph, int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size);
void __device__ sum_1_nAtm(gpu_runtime::GraphT graph, int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size);
void __device__ get_rst_11(int32_t v, int32_t f);
void __device__ get_rst_12(int32_t v, int32_t f);
void __device__ init_feature(int32_t v, int32_t f);

//template <typename EdgeWeightType>
void __device__ gpu_operator_body_0(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
	// Body of the actual operator
	sum_1(graph, dst, src, feat, A, C, Feat_Size);
}

//template <typename EdgeWeightType>
void __device__ gpu_operator_body_0_nAtm(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
	// Body of the actual operator
	sum_1_nAtm(graph, dst, src, feat, A, C, Feat_Size);
}

// template<typename EdgeWeightType>
void __device__ sum_1(gpu_runtime::GraphT graph, int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size) {
    gpu_runtime::writeAdd(&C[((dst * Feat_Size) + feat)], A[((src * Feat_Size) + feat)] / graph.d_get_degree(dst));
}

// template<typename EdgeWeightType>
void __device__ sum_1_nAtm(gpu_runtime::GraphT graph, int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size) {
    C[((dst * Feat_Size) + feat)] += A[((src * Feat_Size) + feat)] / graph.d_get_degree(dst);
}

void __device__ init_feature(int32_t v, int32_t f) {
	int32_t f_size = input_f_size;
	feature_input[f] = (v + f%f_size + 1)/100000.0;
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////
// rewrite get_rst_1

void __device__ get_rst_11(int32_t v, int32_t f) {
    feature_rst_1[f] = feature_input_fc_1[f] + feature_input_out_1[f];
}

void __device__ get_rst_12(int32_t v, int32_t f) {
    float tmp = feature_rst_1[f];
    feature_rst_1[f] = tmp > 0 ? tmp : 0; // get_rst_1 have relu activation
}
////////////////////////////////////////////////////////////////////////////////////////////////////////////

int __host__ main(int argc, char* argv[]) {
    hipError_t cuda_error;
    hipblasStatus_t cublas_error;
    // Initialize CUDA
    cuda_error = hipSetDevice(0);
    if (cuda_error != hipSuccess) {
        cout << "Error: hipSetDevice failed!" << endl;
        return EXIT_FAILURE;
    }

    // Initialize CUBLAS
    const int GPUNUM = 1;
    hipblasHandle_t* cublasHs = new hipblasHandle_t[GPUNUM];
    for (int i = 0; i < GPUNUM; i++) {
        cublas_error = hipblasCreate(&cublasHs[i]);
        if (cublas_error != HIPBLAS_STATUS_SUCCESS) {
            cout << "Error: hipblasCreate failed!" << endl;
            return EXIT_FAILURE;
        }
    }

	__delta_param = 1;
	gpu_runtime::load_graph(__host_edges, argv[1], false);
	hipMemcpyToSymbol(HIP_SYMBOL(edges), &__host_edges, sizeof(__host_edges), 0, hipMemcpyHostToDevice);

    // printf("Graph loaded\n");
    // printf("Graph: %d nodes, %d edges\n", __host_edges.num_vertices, __host_edges.num_edges);

	__host_edges__transposed = gpu_runtime::builtin_transpose(__host_edges);
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(edges__transposed), &__host_edges__transposed, sizeof(__host_edges__transposed), 0, hipMemcpyHostToDevice));

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf("192 CUDA Error: %s\n", hipGetErrorString(cuda_error));
    }

	__host_input_f_size = std::stoi(argv[2]);
	hipMemcpyToSymbol(HIP_SYMBOL(input_f_size), &__host_input_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_hidden_f_size = std::stoi(argv[3]);
	hipMemcpyToSymbol(HIP_SYMBOL(hidden_f_size), &__host_hidden_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_output_f_size = std::stoi(argv[4]);
	hipMemcpyToSymbol(HIP_SYMBOL(output_f_size), &__host_output_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_vector_size = __host_edges__transposed.num_vertices;
	hipMemcpyToSymbol(HIP_SYMBOL(vector_size), &__host_vector_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);

    sz_fin = __host_input_f_size * __host_vector_size;
    sz_fhid = __host_hidden_f_size * __host_vector_size;
    
    sz_w1 = __host_input_f_size * __host_hidden_f_size;
    
    //layer-1
    hipMalloc(&__device_feature_input, sz_fin * sizeof(float));
    hipMemset(__device_feature_input, 0, sz_fin * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input), &__device_feature_input, sizeof(float*), 0);
	__host_feature_input = new float[sz_fin];
	
    hipMalloc(&__device_feature_input_out_1, sz_fhid * sizeof(float));
    hipMemset(__device_feature_input_out_1, 0, sz_fhid * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(feature_input_out_1), &__device_feature_input_out_1, sizeof(float*), 0);
	__host_feature_input_out_1 = new float[sz_fhid];

    hipMalloc(&__device_feature_input_fc_1, sz_fhid * sizeof(float));
    hipMemset(__device_feature_input_fc_1, 0, sz_fhid * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_fc_1), &__device_feature_input_fc_1, sizeof(float*), 0);
    __host_feature_input_fc_1 = new float[sz_fhid];

    hipMalloc(&__device_feature_input_fc_ngh_1, sz_fhid * sizeof(float));
    hipMemset(__device_feature_input_fc_ngh_1, 0, sz_fhid * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_fc_ngh_1), &__device_feature_input_fc_ngh_1, sizeof(float*), 0);
    __host_feature_input_fc_ngh_1 = new float[sz_fhid];

    hipMalloc(&__device_weight_fc_1, sz_w1 * sizeof(float));
    hipMemset(__device_weight_fc_1, 0, sz_w1 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_fc_1), &__device_weight_fc_1, sizeof(float*), 0);
    __host_weight_fc_1 = new float[sz_w1];

    hipMalloc(&__device_weight_fc_ngh_1, sz_w1 * sizeof(float));
    hipMemset(__device_weight_fc_ngh_1, 0, sz_w1 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_fc_ngh_1), &__device_weight_fc_ngh_1, sizeof(float*), 0);
    __host_weight_fc_ngh_1 = new float[sz_w1];

    hipMalloc(&__device_feature_rst_1, sz_fhid * sizeof(float));
    hipMemset(__device_feature_rst_1, 0, sz_fhid * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_rst_1), &__device_feature_rst_1, sizeof(float*), 0);
    __host_feature_rst_1 = new float[sz_fhid];

    // printf("init __host_weight_fc_1:%d %d %d\n", __host_input_f_size, __host_hidden_f_size, __host_hidden_f_size*__host_input_f_size);
    initGPUData(__device_weight_fc_1, sz_w1, 0.001);

    // printf("init __host_weight_fc_ngh_1:%d %d %d\n", __host_input_f_size, __host_hidden_f_size, __host_hidden_f_size*__host_input_f_size);
    initGPUData(__device_weight_fc_ngh_1, sz_w1, 0.002);

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf("300 CUDA Error: %s\n", hipGetErrorString(cuda_error));
    }

    bool run_vf = std::stoi(argv[5]);
    bool run_ef = std::stoi(argv[6]);

    // decide load balance schedule
	gpu_runtime::gnn_load_balance_type lb_c;
    int32_t ef_num_cta=0;
    int32_t ef_feat_size=__host_hidden_f_size;
    int32_t group_size=std::stoi(argv[9]);
    int32_t par_tiling=std::stoi(argv[10]);
    gpu_runtime::uGrapher_init<gpu_operator_body_0, gpu_operator_body_0_nAtm>(
        0, argv[8], &lb_c, __host_edges__transposed, &ef_num_cta, 
        &ef_feat_size, &group_size, &par_tiling);
    
    gpu_runtime::f_vertex_set_apply_host<init_feature>(__host_edges__transposed, __host_input_f_size);
    
    int32_t vf_num_cta[2];
	int32_t vf_feat_size[2];
    vf_feat_size[0] = __host_hidden_f_size;
    vf_feat_size[1] = __host_hidden_f_size;
    gpu_runtime::f_vertex_set_apply_info(__host_edges__transposed, vf_num_cta[0], vf_feat_size[0]);
    gpu_runtime::f_vertex_set_apply_info(__host_edges__transposed, vf_num_cta[1], vf_feat_size[1]);

    float time = 0, time_vf = 0, time_ef = 0;
    float elapsed_time;
    int cnt = 20;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	for (int32_t trail = 0; trail < cnt; trail++) {
        
        // layer-1

        // matmul
        gpu_runtime::matmul_NN(cublasHs, __device_feature_input, __device_weight_fc_1, __device_feature_input_fc_1, __host_vector_size, __host_hidden_f_size, __host_input_f_size);

        gpu_runtime::matmul_NN(cublasHs, __device_feature_input, __device_weight_fc_ngh_1, __device_feature_input_fc_ngh_1, __host_vector_size, __host_hidden_f_size, __host_input_f_size);
        hipDeviceSynchronize();

        // gather
        if (run_ef) hipEventRecord(start);
        {
            gpu_runtime::uGrapher_exec(0, &lb_c, &ef_num_cta, 
                __host_edges__transposed, __device_feature_input_fc_ngh_1, 0, __device_feature_input_out_1, 
                &ef_feat_size, &group_size, &par_tiling);
            // hipDeviceSynchronize();
        }
        if (run_ef) {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed_time, start, stop);
		    time_ef = time_ef + elapsed_time;
        }

        // get_rst
        if (run_vf) hipEventRecord(start);
        {
            gpu_runtime::f_vertex_set_apply_kernel<get_rst_11><<<vf_num_cta[0], CTA_SIZE>>>(__host_edges__transposed, vf_feat_size[0]);
            // hipDeviceSynchronize();
        }
        if (run_vf) {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed_time, start, stop);
		    time_vf = time_vf + elapsed_time;
        }
        if (run_vf) hipEventRecord(start);
        {
            gpu_runtime::f_vertex_set_apply_kernel<get_rst_12><<<vf_num_cta[1], CTA_SIZE>>>(__host_edges__transposed, vf_feat_size[1]);
            // hipDeviceSynchronize();
        }
        if (run_vf) {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed_time, start, stop);
		    time_vf = time_vf + elapsed_time;
        }
    }

    time_vf = time_vf * 1000 / cnt;
    time_ef = time_ef * 1000 / cnt;
    time = time_vf * run_vf + time_ef * run_ef;
    cout << "time: " << time << " msec" << endl;

	// std::cout << (all_time / ((float)1000) * 1000) << std::endl;

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf("Last CUDA Error: %s\n", hipGetErrorString(cuda_error));
    }

    std::fstream fp;
	if (run_vf) {
        fp.open(argv[11], std::ios::out|std::ios::app);
        fp << time_vf;
        fp.close();
    }
    if (run_ef) {
        fp.open(argv[11], std::ios::out|std::ios::app);
        fp << time_ef;
        fp.close();
    }

    //ready to return
    hipFree(__device_feature_input);
    hipFree(__device_feature_input_fc_1);
    hipFree(__device_feature_input_out_1);
    hipFree(__device_feature_input_fc_ngh_1);
    hipFree(__device_feature_rst_1);
    hipFree(__device_weight_fc_1);
    hipFree(__device_weight_fc_ngh_1);

    delete __host_feature_input;
    delete __host_feature_input_fc_1;
    delete __host_feature_input_out_1;
    delete __host_feature_input_fc_ngh_1;
    delete __host_feature_rst_1;
    delete __host_weight_fc_1;
    delete __host_weight_fc_ngh_1;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    for (int i = 0; i < GPUNUM; i++) {
        hipblasDestroy(cublasHs[i]);
    }

    delete cublasHs;

    return 0;
}