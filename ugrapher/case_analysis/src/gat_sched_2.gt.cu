#include "hip/hip_runtime.h"
#include "gpu_intrinsics.h"
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

#define sz_head         8

// edge feature size
int32_t sz_edge_feat_1;      // (model_edge * sz_head)

// leaky_relu_param
#define negative_slope  0.01

int32_t __delta_param;
gpu_runtime::GraphT __device__ edges;
gpu_runtime::GraphT __host_edges;
gpu_runtime::GraphT __device__ edges__transposed;
gpu_runtime::GraphT __host_edges__transposed;

int32_t __device__ input_f_size; 
int32_t __host_input_f_size;

int32_t __device__ hidden_f_size; 
int32_t __host_hidden_f_size;

int32_t __device__ output_f_size; 
int32_t __host_output_f_size;

int32_t __device__ vector_size; 
int32_t __host_vector_size;

int32_t __device__ edge_size;
int32_t __host_edge_size;

int32_t __device__ num_head; 
int32_t __host_num_head;

float __device__ *edge_exp_1;
float *__host_edge_exp_1;
float *__device_edge_exp_1;

// layer1

////////////////////////////////////////////////////////////////////////////////
// rewrite u_add_v_1
void __device__ expf_1(int32_t edge, int32_t feat, float* A, float* C, int Feat_Size);
//template <typename EdgeWeightType>
void __device__ gpu_operator_body_02(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
    // Body of the actual operator
    expf_1(edge, feat, A, C, Feat_Size);
}
void __device__ expf_1(int32_t edge, int32_t feat, float* A, float* C, int Feat_Size) {
    // multi-head
    C[edge * Feat_Size + feat] = __expf(A[edge * Feat_Size + feat]);
}
// rewrite u_add_v_1 end
////////////////////////////////////////////////////////////////////////////////

int __host__ main(int argc, char* argv[]) {
    hipError_t cuda_error;
    hipblasStatus_t cublas_error;
    // Initialize CUDA
    cuda_error = hipSetDevice(0);
    if (cuda_error != hipSuccess) {
        cout << "Error: hipSetDevice failed!" << endl;
        return EXIT_FAILURE;
    }

    // Initialize CUBLAS
    const int GPUNUM = 1;
    hipblasHandle_t* cublasHs = new hipblasHandle_t[GPUNUM];
    for (int i = 0; i < GPUNUM; i++) {
        cublas_error = hipblasCreate(&cublasHs[i]);
        if (cublas_error != HIPBLAS_STATUS_SUCCESS) {
            cout << "Error: hipblasCreate failed!" << endl;
            return EXIT_FAILURE;
        }
    }

	__delta_param = 1;
	gpu_runtime::load_graph(__host_edges, argv[1], false);
	hipMemcpyToSymbol(HIP_SYMBOL(edges), &__host_edges, sizeof(__host_edges), 0, hipMemcpyHostToDevice);

    // printf("Graph loaded\n");
    // printf("Graph: %d nodes, %d edges\n", __host_edges.num_vertices, __host_edges.num_edges);

	__host_edges__transposed = gpu_runtime::builtin_transpose(__host_edges);
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(edges__transposed), &__host_edges__transposed, sizeof(__host_edges__transposed), 0, hipMemcpyHostToDevice));

	__host_input_f_size = std::stoi(argv[2]);
	hipMemcpyToSymbol(HIP_SYMBOL(input_f_size), &__host_input_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_hidden_f_size = std::stoi(argv[3]);
	hipMemcpyToSymbol(HIP_SYMBOL(hidden_f_size), &__host_hidden_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_output_f_size = std::stoi(argv[4]);
	hipMemcpyToSymbol(HIP_SYMBOL(output_f_size), &__host_output_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_vector_size = __host_edges__transposed.num_vertices;
	hipMemcpyToSymbol(HIP_SYMBOL(vector_size), &__host_vector_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_edge_size = __host_edges__transposed.num_edges;
	hipMemcpyToSymbol(HIP_SYMBOL(edge_size), &__host_edge_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_num_head = sz_head;
	hipMemcpyToSymbol(HIP_SYMBOL(num_head), &__host_num_head, sizeof(int32_t), 0, hipMemcpyHostToDevice);

    sz_edge_feat_1 = __host_edge_size * sz_head;

	hipMalloc(&__device_edge_exp_1, sz_edge_feat_1 * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(edge_exp_1), &__device_edge_exp_1, sizeof(float*), 0);
	__host_edge_exp_1 = new float[sz_edge_feat_1];
    
    initGPUData(__device_edge_exp_1, sz_edge_feat_1, 0.001);

    bool run_vf = std::stoi(argv[5]);
    bool run_ef = std::stoi(argv[6]);

    // decide load balance schedule
	gpu_runtime::gnn_load_balance_type lb_c;
    int32_t ef_num_cta=0;
    int32_t ef_feat_size=sz_head;
    int32_t group_size=std::stoi(argv[9]);
    int32_t par_tiling=std::stoi(argv[10]);
    gpu_runtime::uGrapher_init<gpu_operator_body_02, gpu_operator_body_02>(
        0, argv[8], &lb_c, __host_edges__transposed, &ef_num_cta, 
        &ef_feat_size, &group_size, &par_tiling);
    
    float time = 0, time_vf = 0, time_ef = 0;
    float elapsed_time;
    int cnt = 20;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	for (int32_t trail = 0; trail < cnt; trail++) {

        //layer_1
        if (run_ef) hipEventRecord(start);
        {
			gpu_runtime::uGrapher_exec(0, &lb_c, &ef_num_cta, 
                __host_edges__transposed, __device_edge_exp_1, 0, __device_edge_exp_1, 
                &ef_feat_size, &group_size, &par_tiling);
            // hipDeviceSynchronize();
		}
        if (run_ef) {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed_time, start, stop);
		    time_ef = time_ef + elapsed_time;
        }
	}
    
    time_vf = time_vf * 1000 / cnt;
    time_ef = time_ef * 1000 / cnt;
    time = time_vf * run_vf + time_ef * run_ef;
    cout << "time: " << time << " msec" << endl;

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf(" 600 CUDA Error: %s\n", hipGetErrorString(cuda_error));
        return 0;
    }

    std::fstream fp;
    if (run_vf) {
        fp.open(argv[11], std::ios::out|std::ios::app);
        fp << time_vf;
        fp.close();
    }
    if (run_ef) {
        fp.open(argv[11], std::ios::out|std::ios::app);
        fp << time_ef;
        fp.close();
    }

    //free
    CUDA_CALL(hipFree(__device_edge_exp_1));

    delete __host_edge_exp_1;

    // cublas free
    for (int i = 0; i < GPUNUM; i++) {
		hipblasDestroy(cublasHs[i]);
	}

	delete cublasHs;
}