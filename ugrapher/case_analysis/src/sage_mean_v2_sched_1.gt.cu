// full-batch
// cora
// aggregation op = sum/pool/lstm/mean
// this file is for mean aggregation
#include "gpu_intrinsics.h"
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

// total feature size
int32_t sz_fhid;		// (model_fhid * model_vertex)
int32_t sz_fout;		// (model_fout * model_vertex)

// weight size
int32_t sz_w2;          // (model_fhid * model_fout)

int32_t __delta_param;
gpu_runtime::GraphT __device__ edges;
gpu_runtime::GraphT __host_edges;
gpu_runtime::GraphT __device__ edges__transposed;
gpu_runtime::GraphT __host_edges__transposed;

int32_t __device__ M; 
int32_t __host_M;

int32_t __device__ input_f_size; 
int32_t __host_input_f_size;

int32_t __device__ hidden_f_size; 
int32_t __host_hidden_f_size;

int32_t __device__ output_f_size; 
int32_t __host_output_f_size;

int32_t __device__ vector_size; 
int32_t __host_vector_size;

float __device__ *feature_input_out_2;
float *__host_feature_input_out_2;
float *__device_feature_input_out_2;

float __device__ *feature_input_fc_2;
float *__host_feature_input_fc_2;
float *__device_feature_input_fc_2;

float __device__ *feature_input_fc_ngh_2;
float *__host_feature_input_fc_ngh_2;
float *__device_feature_input_fc_ngh_2;

float __device__ *feature_rst_1;
float *__host_feature_rst_1;
float *__device_feature_rst_1;

float __device__ *feature_rst_2;
float *__host_feature_rst_2;
float *__device_feature_rst_2;

float __device__ *weight_fc_2;
float *__host_weight_fc_2;
float *__device_weight_fc_2;

float __device__ *weight_fc_ngh_2;
float *__host_weight_fc_ngh_2;
float *__device_weight_fc_ngh_2;

void __device__ sum_2(gpu_runtime::GraphT graph, int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size);
void __device__ sum_2_nAtm(gpu_runtime::GraphT graph, int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size);
void __device__ get_rst_2(int32_t v, int32_t f);
void __device__ init_feature(int32_t v, int32_t f);

//template <typename EdgeWeightType>
void __device__ gpu_operator_body_1(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
	// Body of the actual operator
	sum_2(graph, dst, src, feat, A, C, Feat_Size);
}

//template <typename EdgeWeightType>
void __device__ gpu_operator_body_1_nAtm(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
	// Body of the actual operator
	sum_2_nAtm(graph, dst, src, feat, A, C, Feat_Size);
}

// template<typename EdgeWeightType>
void __device__ sum_2(gpu_runtime::GraphT graph, int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size) {
    gpu_runtime::writeAdd(&C[((dst * Feat_Size) + feat)], A[((src * Feat_Size) + feat)] / graph.d_get_degree(dst));
}

// template<typename EdgeWeightType>
void __device__ sum_2_nAtm(gpu_runtime::GraphT graph, int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size) {
    C[((dst * Feat_Size) + feat)] += A[((src * Feat_Size) + feat)] / graph.d_get_degree(dst);
}

void __device__ init_feature(int32_t v, int32_t f) {
	int32_t f_size = hidden_f_size;
	feature_rst_1[f] = (v + f%f_size + 1)/100000.0;
}

void __device__ get_rst_2(int32_t v, int32_t f) {
    feature_rst_2[f] = feature_input_fc_2[f] + feature_input_fc_ngh_2[f]; //get_rst_2 not need activation
}

int __host__ main(int argc, char* argv[]) {
    hipError_t cuda_error;
    hipblasStatus_t cublas_error;
    // Initialize CUDA
    cuda_error = hipSetDevice(0);
    if (cuda_error != hipSuccess) {
        cout << "Error: hipSetDevice failed!" << endl;
        return EXIT_FAILURE;
    }

    // Initialize CUBLAS
    const int GPUNUM = 1;
    hipblasHandle_t* cublasHs = new hipblasHandle_t[GPUNUM];
    for (int i = 0; i < GPUNUM; i++) {
        cublas_error = hipblasCreate(&cublasHs[i]);
        if (cublas_error != HIPBLAS_STATUS_SUCCESS) {
            cout << "Error: hipblasCreate failed!" << endl;
            return EXIT_FAILURE;
        }
    }

	__delta_param = 1;
	gpu_runtime::load_graph(__host_edges, argv[1], false);
	hipMemcpyToSymbol(HIP_SYMBOL(edges), &__host_edges, sizeof(__host_edges), 0, hipMemcpyHostToDevice);

    // printf("Graph loaded\n");
    // printf("Graph: %d nodes, %d edges\n", __host_edges.num_vertices, __host_edges.num_edges);

	__host_edges__transposed = gpu_runtime::builtin_transpose(__host_edges);
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(edges__transposed), &__host_edges__transposed, sizeof(__host_edges__transposed), 0, hipMemcpyHostToDevice));

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf("192 CUDA Error: %s\n", hipGetErrorString(cuda_error));
    }

	__host_input_f_size = std::stoi(argv[2]);
	hipMemcpyToSymbol(HIP_SYMBOL(input_f_size), &__host_input_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_hidden_f_size = std::stoi(argv[3]);
	hipMemcpyToSymbol(HIP_SYMBOL(hidden_f_size), &__host_hidden_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_output_f_size = std::stoi(argv[4]);
	hipMemcpyToSymbol(HIP_SYMBOL(output_f_size), &__host_output_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_vector_size = __host_edges__transposed.num_vertices;
	hipMemcpyToSymbol(HIP_SYMBOL(vector_size), &__host_vector_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);

    sz_fhid = __host_hidden_f_size * __host_vector_size;
    sz_fout = __host_output_f_size * __host_vector_size;
    
    sz_w2 = __host_hidden_f_size * __host_output_f_size;
    
    //layer-1
    hipMalloc(&__device_feature_rst_1, sz_fhid * sizeof(float));
    hipMemset(__device_feature_rst_1, 0, sz_fhid * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_rst_1), &__device_feature_rst_1, sizeof(float*), 0);
    __host_feature_rst_1 = new float[sz_fhid];

    //layer-2
    hipMalloc(&__device_feature_input_out_2, sz_fhid * sizeof(float));
    hipMemset(__device_feature_input_out_2, 0, sz_fhid * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_out_2), &__device_feature_input_out_2, sizeof(float*), 0);
    __host_feature_input_out_2 = new float[sz_fhid];

    hipMalloc(&__device_feature_input_fc_2, sz_fout * sizeof(float));
    hipMemset(__device_feature_input_fc_2, 0, sz_fout * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_fc_2), &__device_feature_input_fc_2, sizeof(float*), 0);
    __host_feature_input_fc_2 = new float[sz_fout];

    hipMalloc(&__device_feature_input_fc_ngh_2, sz_fout * sizeof(float));
    hipMemset(__device_feature_input_fc_ngh_2, 0, sz_fout * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_fc_ngh_2), &__device_feature_input_fc_ngh_2, sizeof(float*), 0);
    __host_feature_input_fc_ngh_2 = new float[sz_fout];

    hipMalloc(&__device_weight_fc_2, sz_w2 * sizeof(float));
    hipMemset(__device_weight_fc_2, 0, sz_w2 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_fc_2), &__device_weight_fc_2, sizeof(float*), 0);
    __host_weight_fc_2 = new float[sz_w2];

    hipMalloc(&__device_weight_fc_ngh_2, sz_w2 * sizeof(float));
    hipMemset(__device_weight_fc_ngh_2, 0, sz_w2 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_fc_ngh_2), &__device_weight_fc_ngh_2, sizeof(float*), 0);
    __host_weight_fc_ngh_2 = new float[sz_w2];

    hipMalloc(&__device_feature_rst_2, sz_fout * sizeof(float));
    hipMemset(__device_feature_rst_2, 0, sz_fout * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_rst_2), &__device_feature_rst_2, sizeof(float*), 0);
    __host_feature_rst_2 = new float[sz_fout];

    // printf("init __host_weight_fc_2:%d %d %d\n", __host_hidden_f_size, __host_output_f_size, __host_output_f_size*__host_hidden_f_size);
    initGPUData(__device_weight_fc_2, sz_w2, 0.003);

    // printf("init __host_weight_fc_ngh_2:%d %d %d\n", __host_hidden_f_size, __host_output_f_size, __host_output_f_size*__host_hidden_f_size);
    initGPUData(__device_weight_fc_ngh_2, sz_w2, 0.004);

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf("300 CUDA Error: %s\n", hipGetErrorString(cuda_error));
    }

    bool run_vf = std::stoi(argv[5]);
    bool run_ef = std::stoi(argv[6]);

    // decide load balance schedule
	gpu_runtime::gnn_load_balance_type lb_c;
    int32_t ef_num_cta=0;
    int32_t ef_feat_size=__host_hidden_f_size;
    int32_t group_size=std::stoi(argv[9]);
    int32_t par_tiling=std::stoi(argv[10]);
    gpu_runtime::uGrapher_init<gpu_operator_body_1, gpu_operator_body_1_nAtm>(
        0, argv[8], &lb_c, __host_edges__transposed, &ef_num_cta, 
        &ef_feat_size, &group_size, &par_tiling);
    
    gpu_runtime::f_vertex_set_apply_host<init_feature>(__host_edges__transposed, __host_hidden_f_size);
    
    int32_t vf_num_cta;
	int32_t vf_feat_size = __host_output_f_size;
    gpu_runtime::f_vertex_set_apply_info(__host_edges__transposed, vf_num_cta, vf_feat_size);

    float time = 0, time_vf = 0, time_ef = 0;
    float elapsed_time;
    int cnt = 20;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

	for (int32_t trail = 0; trail < cnt; trail++) {
        // layer-2
        // matmul
        gpu_runtime::matmul_NN(cublasHs, __device_feature_rst_1, __device_weight_fc_2, __device_feature_input_fc_2, __host_vector_size, __host_output_f_size, __host_hidden_f_size);
        hipDeviceSynchronize();

        // gather
        if (run_ef) hipEventRecord(start);
        {
            gpu_runtime::uGrapher_exec(0, &lb_c, &ef_num_cta, 
                __host_edges__transposed, __device_feature_rst_1, 0, __device_feature_input_out_2, 
                &ef_feat_size, &group_size, &par_tiling);
            // hipDeviceSynchronize();
        }
        if (run_ef) {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed_time, start, stop);
		    time_ef = time_ef + elapsed_time;
        }

        // mean
        
        gpu_runtime::matmul_NN(cublasHs, __device_feature_input_out_2, __device_weight_fc_ngh_2, __device_feature_input_fc_ngh_2, __host_vector_size, __host_output_f_size, __host_hidden_f_size);
        hipDeviceSynchronize();

        // get_rst
        if (run_vf) hipEventRecord(start);
        {
            gpu_runtime::f_vertex_set_apply_kernel<get_rst_2><<<vf_num_cta, CTA_SIZE>>>(__host_edges__transposed, vf_feat_size);
            // hipDeviceSynchronize();
        }
        if (run_vf) {
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&elapsed_time, start, stop);
		    time_vf = time_vf + elapsed_time;
        }
    }

    time_vf = time_vf * 1000 / cnt;
    time_ef = time_ef * 1000 / cnt;
    time = time_vf * run_vf + time_ef * run_ef;
    cout << "time: " << time << " msec" << endl;

	// std::cout << (all_time / ((float)1000) * 1000) << std::endl;

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf("Last CUDA Error: %s\n", hipGetErrorString(cuda_error));
    }

    std::fstream fp;
	if (run_vf) {
        fp.open(argv[11], std::ios::out|std::ios::app);
        fp << time_vf;
        fp.close();
    }
    if (run_ef) {
        fp.open(argv[11], std::ios::out|std::ios::app);
        fp << time_ef;
        fp.close();
    }

    //ready to return
    hipFree(__device_feature_rst_1);
    hipFree(__device_feature_input_fc_2);
    hipFree(__device_feature_input_out_2);
    hipFree(__device_feature_input_fc_ngh_2);
    hipFree(__device_feature_rst_2);
    hipFree(__device_weight_fc_2);
    hipFree(__device_weight_fc_ngh_2);

    delete __host_feature_rst_1;
    delete __host_feature_input_fc_2;
    delete __host_feature_input_out_2;
    delete __host_feature_input_fc_ngh_2;
    delete __host_feature_rst_2;
    delete __host_weight_fc_2;
    delete __host_weight_fc_ngh_2;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    for (int i = 0; i < GPUNUM; i++) {
        hipblasDestroy(cublasHs[i]);
    }

    delete cublasHs;

    return 0;
}