#include "hip/hip_runtime.h"
#include "gpu_intrinsics.h"
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

// total feature size
int32_t sz_fin;		    // (model_fin  * model_vertex)
int32_t sz_fhid;		// (model_fhid * model_vertex)
int32_t sz_fout;		// (model_fout * model_vertex)

// weight size
int32_t sz_w1;          // (model_fin * model_fhid)
int32_t sz_w2;          // (model_fhid * model_fout)

// norm weight
// #define sz_nw       (size_t(model_vertex) * size_t(model_vertex))   
int32_t sz_edge_nw;     // (size_t(model_edge))

int32_t __delta_param;
gpu_runtime::GraphT __device__ edges;
gpu_runtime::GraphT __host_edges;
gpu_runtime::GraphT __device__ edges__transposed;
gpu_runtime::GraphT __host_edges__transposed;
// gpu_runtime::GraphT<char> __device__ edges;
// gpu_runtime::GraphT<char> __host_edges;
// gpu_runtime::GraphT<char> __device__ edges__transposed;
// gpu_runtime::GraphT<char> __host_edges__transposed;

int32_t __device__ input_f_size; 
int32_t __host_input_f_size;

int32_t __device__ hidden_f_size; 
int32_t __host_hidden_f_size;

int32_t __device__ output_f_size; 
int32_t __host_output_f_size;

int32_t __device__ vector_size; 
int32_t __host_vector_size;

int32_t __device__ edge_size;
int32_t __host_edge_size;

float __device__ *feature_input;
float *__host_feature_input;
float *__device_feature_input;

float __device__ *feature_input_fc_1;
float *__host_feature_input_fc_1;
float *__device_feature_input_fc_1;

float __device__ *feature_input_fc_2;
float *__host_feature_input_fc_2;
float *__device_feature_input_fc_2;

float __device__ *feature_input_out_1;
float *__host_feature_input_out_1;
float *__device_feature_input_out_1;

float __device__ *feature_input_out_2;
float *__host_feature_input_out_2;
float *__device_feature_input_out_2;

float __device__ *feature_rst_1;
float *__host_feature_rst_1;
float *__device_feature_rst_1;

float __device__ *feature_rst_2;
float *__host_feature_rst_2;
float *__device_feature_rst_2;

float __device__ *weight_fc_1;
float *__host_weight_fc_1;
float *__device_weight_fc_1;

float __device__ *weight_fc_2;
float *__host_weight_fc_2;
float *__device_weight_fc_2;

float __device__ *weight_norm;
float *__host_weight_norm;
float *__device_weight_norm;

void __device__ sum_1(int32_t src, int32_t dst, int32_t edge, int32_t feat, float* A, float* B, float* C, int Feat_Size);
void __device__ sum_1_nAtm(int32_t src, int32_t dst, int32_t edge, int32_t feat, float* A, float* B, float* C, int Feat_Size);
void __device__ sum_2(int32_t src, int32_t dst, int32_t edge, int32_t feat, float* A, float* B, float* C, int Feat_Size);
void __device__ sum_2_nAtm(int32_t src, int32_t dst, int32_t edge, int32_t feat, float* A, float* B, float* C, int Feat_Size);

void __device__ get_rst_1(int32_t v, int32_t f);
void __device__ get_rst_2(int32_t v, int32_t f);
void __device__ init_feature(int32_t v, int32_t f);

//template <typename EdgeWeightType>
void __device__ gpu_operator_body_0(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
	// Body of the actual operator
	sum_1(dst, src, edge, feat, A, B, C, Feat_Size);
}

//template <typename EdgeWeightType>
void __device__ gpu_operator_body_0_nAtm(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
	// Body of the actual operator
	sum_1_nAtm(dst, src, edge, feat, A, B, C, Feat_Size);
}

//template <typename EdgeWeightType>
void __device__ gpu_operator_body_1(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
	// Body of the actual operator
	sum_2(dst, src, edge, feat, A, B, C, Feat_Size);
}

//template <typename EdgeWeightType>
void __device__ gpu_operator_body_1_nAtm(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
	// Body of the actual operator
	sum_2_nAtm(dst, src, edge, feat, A, B, C, Feat_Size);
}

void __device__ sum_1(int32_t src, int32_t dst, int32_t edge, int32_t feat, float* A, float* B, float* C, int Feat_Size) {
    gpu_runtime::writeAdd(&C[((dst * Feat_Size) + feat)], A[((src * Feat_Size) + feat)] * B[edge]);
}

void __device__ sum_1_nAtm(int32_t src, int32_t dst, int32_t edge, int32_t feat, float* A, float* B, float* C, int Feat_Size) {
    C[((dst * Feat_Size) + feat)] += A[((src * Feat_Size) + feat)] * B[edge];
}

void __device__ sum_2(int32_t src, int32_t dst, int32_t edge, int32_t feat, float* A, float* B, float* C, int Feat_Size) {
    gpu_runtime::writeAdd(&C[((dst * Feat_Size) + feat)], A[((src * Feat_Size) + feat)] * B[edge]);
}

void __device__ sum_2_nAtm(int32_t src, int32_t dst, int32_t edge, int32_t feat, float* A, float* B, float* C, int Feat_Size) {
    C[((dst * Feat_Size) + feat)] += A[((src * Feat_Size) + feat)] * B[edge];
}

void __device__ init_feature(int32_t v, int32_t f) {
	int32_t f_size = input_f_size;
	feature_input[f] = (v + f % f_size + 1) / 10000.0;
}

void __device__ get_rst_1(int32_t v, int32_t f) {
    feature_rst_1[f] = (feature_input_out_1[f] > 0) ? feature_input_out_1[f] : 0;
}

void __device__ get_rst_2(int32_t v, int32_t f) {
    feature_rst_2[f] = (feature_input_out_2[f] > 0) ? feature_input_out_2[f] : 0;
}

__global__ void initWeightNorm_ker(float *data, size_t numElements) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numElements) {
        size_t src_id, dst_id;
        src_id = edges__transposed.d_edge_src[tid];
        dst_id = edges__transposed.d_edge_dst[tid];

        size_t src_deg, dst_deg;
        src_deg = edges__transposed.d_get_degree(src_id);
        dst_deg = edges__transposed.d_get_degree(dst_id);
        data[tid] = pow(src_deg, -0.5) * pow(dst_deg, -0.5);
    }
 }

// used to init norm weight in GCN
void __host__ initWeightNorm(float * weight_norm) {
    dim3 gridDim;
    dim3 blockDim;
 
    blockDim.x = 1024;
    size_t numElements = sz_edge_nw;

    gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;
 
    initWeightNorm_ker <<< gridDim, blockDim >>> (weight_norm, numElements);
}

int __host__ main(int argc, char* argv[]) {
    hipError_t cuda_error;
    hipblasStatus_t cublas_error;
    // Initialize CUDA
    cuda_error = hipSetDevice(0);
    if (cuda_error != hipSuccess) {
        cout << "Error: hipSetDevice failed!" << endl;
        return EXIT_FAILURE;
    }

    // Initialize CUBLAS
    const int GPUNUM = 1;
    hipblasHandle_t* cublasHs = new hipblasHandle_t[GPUNUM];
    for (int i = 0; i < GPUNUM; i++) {
        cublas_error = hipblasCreate(&cublasHs[i]);
        if (cublas_error != HIPBLAS_STATUS_SUCCESS) {
            cout << "Error: hipblasCreate failed!" << endl;
            return EXIT_FAILURE;
        }
    }

    __delta_param = 1;
	gpu_runtime::load_graph(__host_edges, argv[1], false);
	hipMemcpyToSymbol(HIP_SYMBOL(edges), &__host_edges, sizeof(__host_edges), 0, hipMemcpyHostToDevice);

    // printf("Graph loaded\n");
    // printf("Graph: %d nodes, %d edges\n", __host_edges.num_vertices, __host_edges.num_edges);

	__host_edges__transposed = gpu_runtime::builtin_transpose(__host_edges);
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(edges__transposed), &__host_edges__transposed, sizeof(__host_edges__transposed), 0, hipMemcpyHostToDevice));

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf("172 CUDA Error: %s\n", hipGetErrorString(cuda_error));
    }

    __host_input_f_size = std::stoi(argv[2]);
	hipMemcpyToSymbol(HIP_SYMBOL(input_f_size), &__host_input_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_hidden_f_size = std::stoi(argv[3]);
	hipMemcpyToSymbol(HIP_SYMBOL(hidden_f_size), &__host_hidden_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_output_f_size = std::stoi(argv[4]);
	hipMemcpyToSymbol(HIP_SYMBOL(output_f_size), &__host_output_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_vector_size = __host_edges__transposed.num_vertices;
	hipMemcpyToSymbol(HIP_SYMBOL(vector_size), &__host_vector_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
    __host_edge_size = __host_edges__transposed.num_edges;
	hipMemcpyToSymbol(HIP_SYMBOL(edge_size), &__host_edge_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	
    sz_fin = __host_input_f_size * __host_vector_size;
    sz_fhid = __host_hidden_f_size * __host_vector_size;
    sz_fout = __host_output_f_size * __host_vector_size;

    sz_w1 = __host_input_f_size * __host_hidden_f_size;
    sz_w2 = __host_hidden_f_size * __host_output_f_size;

    sz_edge_nw = __host_edge_size;

    //layer-1
    hipMalloc(&__device_feature_input, sz_fin * sizeof(float));
    hipMemset(__device_feature_input, 0, sz_fin * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input), &__device_feature_input, sizeof(float*), 0);
	__host_feature_input = new float[sz_fin];
	
    hipMalloc(&__device_feature_input_fc_1, sz_fhid * sizeof(float));
    hipMemset(__device_feature_input_fc_1, 0, sz_fhid * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_fc_1), &__device_feature_input_fc_1, sizeof(float*), 0);
    __host_feature_input_fc_1 = new float[sz_fhid];

    hipMalloc(&__device_feature_input_out_1, sz_fhid * sizeof(float));
    hipMemset(__device_feature_input_out_1, 0, sz_fhid * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(feature_input_out_1), &__device_feature_input_out_1, sizeof(float*), 0);
	__host_feature_input_out_1 = new float[sz_fhid];

    hipMalloc(&__device_feature_rst_1, sz_fhid * sizeof(float));
    hipMemset(__device_feature_rst_1, 0, sz_fhid * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_rst_1), &__device_feature_rst_1, sizeof(float*), 0);
    __host_feature_rst_1 = new float[sz_fhid];

    hipMalloc(&__device_weight_fc_1, sz_w1 * sizeof(float));
    hipMemset(__device_weight_fc_1, 0, sz_w1 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_fc_1), &__device_weight_fc_1, sizeof(float*), 0);
    __host_weight_fc_1 = new float[sz_w1];

    //layer-2
    hipMalloc(&__device_feature_input_fc_2, sz_fout * sizeof(float));
    hipMemset(__device_feature_input_fc_2, 0, sz_fout * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_fc_2), &__device_feature_input_fc_2, sizeof(float*), 0);
    __host_feature_input_fc_2 = new float[sz_fout];

    hipMalloc(&__device_feature_input_out_2, sz_fout * sizeof(float));
    hipMemset(__device_feature_input_out_2, 0, sz_fout * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_out_2), &__device_feature_input_out_2, sizeof(float*), 0);
    __host_feature_input_out_2 = new float[sz_fout];

    hipMalloc(&__device_feature_rst_2, sz_fout * sizeof(float));
    hipMemset(__device_feature_rst_2, 0, sz_fout * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_rst_2), &__device_feature_rst_2, sizeof(float*), 0);
    __host_feature_rst_2 = new float[sz_fout];

    hipMalloc(&__device_weight_fc_2, sz_w2 * sizeof(float));
    hipMemset(__device_weight_fc_2, 0, sz_w2 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_fc_2), &__device_weight_fc_2, sizeof(float*), 0);
    __host_weight_fc_2 = new float[sz_w2];

    hipMalloc(&__device_weight_norm, sz_edge_nw * sizeof(float));
    cuda_error = hipGetLastError();
    if (cuda_error != hipSuccess) {
        printf("295 CUDA Error: %s\n", hipGetErrorString(cuda_error));
        return -1;
    }
    hipMemset(__device_weight_norm, 0, sz_edge_nw * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_norm), &__device_weight_norm, sizeof(float*), 0);
    __host_weight_norm = new float[sz_edge_nw];

    initGPUData(__device_weight_fc_1, sz_w1, 0.001);
    initGPUData(__device_weight_fc_2, sz_w2, 0.002);
    // initGPUData(__device_weight_norm, __host_vector_size * __host_vector_size, 0.3);
    initWeightNorm(__device_weight_norm);

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf("321 CUDA Error: %s\n", hipGetErrorString(cuda_error));
        return -1;
    }

    // decide load balance schedule
	gpu_runtime::gnn_load_balance_type lb_c[2];
    int32_t ef_num_cta[2]={0,0};
    int32_t ef_feat_size[2];
    ef_feat_size[0] = __host_hidden_f_size;
    ef_feat_size[1] = __host_output_f_size;
    int32_t group_size[2]={std::stoi(argv[9]),std::stoi(argv[13])};
    int32_t par_tiling[2]={std::stoi(argv[10]),std::stoi(argv[14])};
    gpu_runtime::uGrapher_init<gpu_operator_body_0, gpu_operator_body_0_nAtm>(
        0, argv[8], lb_c, __host_edges__transposed, ef_num_cta, 
        ef_feat_size, group_size, par_tiling);
    gpu_runtime::uGrapher_init<gpu_operator_body_1, gpu_operator_body_1_nAtm>(
        1, argv[12], lb_c, __host_edges__transposed, ef_num_cta, 
        ef_feat_size, group_size, par_tiling);
    
    gpu_runtime::f_vertex_set_apply_host<init_feature>(__host_edges__transposed, __host_input_f_size);
    
    int32_t vf_num_cta[2];
	int32_t vf_feat_size[2];
    vf_feat_size[0] = __host_hidden_f_size;
    vf_feat_size[1] = __host_output_f_size;
    gpu_runtime::f_vertex_set_apply_info(__host_edges__transposed, vf_num_cta[0], vf_feat_size[0]);
    gpu_runtime::f_vertex_set_apply_info(__host_edges__transposed, vf_num_cta[1], vf_feat_size[1]);
    
    float time = 0;
    int cnt = 20;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

	for (int32_t trail = 0; trail < cnt; trail++) {
        
        //layer-1
        gpu_runtime::matmul_NN(cublasHs, __device_feature_input, __device_weight_fc_1, __device_feature_input_fc_1, __host_vector_size, __host_hidden_f_size, __host_input_f_size);

        // gather
        {
            gpu_runtime::uGrapher_exec(0, lb_c, ef_num_cta, 
                __host_edges__transposed, __device_feature_input_fc_1, __device_weight_norm, __device_feature_input_out_1, 
                ef_feat_size, group_size, par_tiling);
            // hipDeviceSynchronize();
        }

        // get_rst
        {
            gpu_runtime::f_vertex_set_apply_kernel<get_rst_1><<<vf_num_cta[0], CTA_SIZE>>>(__host_edges__transposed, vf_feat_size[0]);
            // hipDeviceSynchronize();
        }

        ////////////////////////////////////////////////////////////////////////////////
        // layer-2
        gpu_runtime::matmul_NN(cublasHs, __device_feature_rst_1, __device_weight_fc_2, __device_feature_input_fc_2, __host_vector_size, __host_output_f_size, __host_hidden_f_size);

        // gather
        {
            gpu_runtime::uGrapher_exec(1, lb_c, ef_num_cta, 
                __host_edges__transposed, __device_feature_input_fc_2, __device_weight_norm, __device_feature_input_out_2, 
                ef_feat_size, group_size, par_tiling);
            // hipDeviceSynchronize();
        }

        // get_rst
        {   
            gpu_runtime::f_vertex_set_apply_kernel<get_rst_2><<<vf_num_cta[1], CTA_SIZE>>>(__host_edges__transposed, vf_feat_size[1]);
            // hipDeviceSynchronize();
	    }
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    time = time / cnt;
    cout << "time: " << time << " msec" << endl;

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf("Last CUDA Error: %s\n", hipGetErrorString(cuda_error));
        return -1;
    }

    std::fstream fp;
	fp.open(argv[15], std::ios::out|std::ios::app);
	fp << time << std::endl;
	fp.close();

    //ready to return
    hipFree(__device_feature_input);
    hipFree(__device_feature_input_fc_1);
    hipFree(__device_feature_input_out_1);
    hipFree(__device_feature_rst_1);
    hipFree(__device_feature_input_fc_2);
    hipFree(__device_feature_input_out_2);
    hipFree(__device_feature_rst_2);
    hipFree(__device_weight_fc_1);
    hipFree(__device_weight_fc_2);

    delete __host_feature_input;
    delete __host_feature_input_fc_1;
    delete __host_feature_input_out_1;
    delete __host_feature_rst_1;
    delete __host_feature_input_fc_2;
    delete __host_feature_input_out_2;
    delete __host_feature_rst_2;
    delete __host_weight_fc_1;
    delete __host_weight_fc_2;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    for (int i = 0; i < GPUNUM; i++) {
		hipblasDestroy(cublasHs[i]);
	}

	delete cublasHs;

    return 0;
}