// full-batch
// aggregation op = sum/pool/lstm/mean
// this file is for sum aggregation
#include "gpu_intrinsics.h"
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

// total feature size
int32_t sz_fin;		    // (model_fin  * model_vertex)
int32_t sz_fhid;		// (model_fhid * model_vertex)
int32_t sz_fout;		// (model_fout * model_vertex)

// weight size
int32_t sz_w1;       // (model_fin * model_fhid)
int32_t sz_w2;       // (model_fhid * model_fout)

int32_t sz_w_pool1;  // (model_fin * model_fin)
int32_t sz_w_pool2;  // (model_fhid * model_fhid)


int32_t __delta_param;
gpu_runtime::GraphT __device__ edges;
gpu_runtime::GraphT __host_edges;
gpu_runtime::GraphT __device__ edges__transposed;
gpu_runtime::GraphT __host_edges__transposed;

int32_t __device__ M; 
int32_t __host_M;

int32_t __device__ input_f_size; 
int32_t __host_input_f_size;

int32_t __device__ hidden_f_size; 
int32_t __host_hidden_f_size;

int32_t __device__ output_f_size; 
int32_t __host_output_f_size;

int32_t __device__ vector_size; 
int32_t __host_vector_size;

float __device__ *feature_input;
float *__host_feature_input;
float *__device_feature_input;

float __device__ *feature_input_pool_1;
float *__host_feature_input_pool_1;
float *__device_feature_input_pool_1;

float __device__ *feature_input_out_1;
float *__host_feature_input_out_1;
float *__device_feature_input_out_1;

float __device__ *feature_input_fc_pool_1;
float *__host_feature_input_fc_pool_1;
float *__device_feature_input_fc_pool_1;

float __device__ *feature_input_fc_self_1;
float *__host_feature_input_fc_self_1;
float *__device_feature_input_fc_self_1;

float __device__ *feature_input_fc_ngh_1;
float *__host_feature_input_fc_ngh_1;
float *__device_feature_input_fc_ngh_1;

float __device__ *feature_rst_1;
float *__host_feature_rst_1;
float *__device_feature_rst_1;

float __device__ *weight_fc_ngh_1;
float *__host_weight_fc_ngh_1;
float *__device_weight_fc_ngh_1;

float __device__ *weight_fc_self_1;
float *__host_weight_fc_self_1;
float *__device_weight_fc_self_1;

float __device__ *weight_pool_1;
float *__host_weight_pool_1;
float *__device_weight_pool_1;

float __device__ *feature_input_pool_2;
float *__host_feature_input_pool_2;
float *__device_feature_input_pool_2;

float __device__ *feature_input_out_2;
float *__host_feature_input_out_2;
float *__device_feature_input_out_2;

float __device__ *feature_input_fc_self_2;
float *__host_feature_input_fc_self_2;
float *__device_feature_input_fc_self_2;

float __device__ *feature_input_fc_ngh_2;
float *__host_feature_input_fc_ngh_2;
float *__device_feature_input_fc_ngh_2;

float __device__ *feature_input_fc_pool_2;
float *__host_feature_input_fc_pool_2;
float *__device_feature_input_fc_pool_2;

float __device__ *feature_rst_2;
float *__host_feature_rst_2;
float *__device_feature_rst_2;

float __device__ *weight_fc_ngh_2;
float *__host_weight_fc_ngh_2;
float *__device_weight_fc_ngh_2;

float __device__ *weight_fc_self_2;
float *__host_weight_fc_self_2;
float *__device_weight_fc_self_2;

float __device__ *weight_pool_2;
float *__host_weight_pool_2;
float *__device_weight_pool_2;

void __device__ max_1(int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size);
void __device__ max_1_nAtm(int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size);
void __device__ get_rst_1(int32_t v, int32_t f);
void __device__ pool_relu_1(int32_t v, int32_t f);
void __device__ max_2(int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size);
void __device__ max_2_nAtm(int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size);
void __device__ get_rst_2(int32_t v, int32_t f);
void __device__ pool_relu_2(int32_t v, int32_t f);
void __device__ init_feature(int32_t v, int32_t f);

//template <typename EdgeWeightType>
void __device__ gpu_operator_body_0(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
	// Body of the actual operator
	max_1(dst, src, feat, A, C, Feat_Size);
}

//template <typename EdgeWeightType>
void __device__ gpu_operator_body_0_nAtm(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
	// Body of the actual operator
	max_1_nAtm(dst, src, feat, A, C, Feat_Size);
}

void __device__ max_1(int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size) {
    MyatomicMax(&C[((dst * Feat_Size) + feat)], A[((src * Feat_Size) + feat)]);
}

void __device__ max_1_nAtm(int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size) {
    float tmp_d = C[((dst * Feat_Size) + feat)];
    float tmp_s = A[((src * Feat_Size) + feat)];
    C[((dst * Feat_Size) + feat)] = tmp_d >= tmp_s ? tmp_d : tmp_s;
}

void __device__ init_feature(int32_t v, int32_t f) {
	int32_t f_size = input_f_size;
	feature_input[f] = (v + f % f_size + 1) / 100000.0;
}

void __device__ get_rst_1(int32_t v, int32_t f) {
    float tmp;
    tmp = feature_input_fc_self_1[f] + feature_input_fc_ngh_1[f];
    feature_rst_1[f] = tmp > 0 ? tmp : 0;
}

void __device__ pool_relu_1(int32_t v, int32_t f) {
    feature_input_pool_1[f] = feature_input_fc_pool_1[f] > 0 ? feature_input_fc_pool_1[f] : 0;
}

//template <typename EdgeWeightType>
void __device__ gpu_operator_body_1(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
	// Body of the actual operator
	max_2(dst, src, feat, A, C, Feat_Size);
}

//template <typename EdgeWeightType>
void __device__ gpu_operator_body_1_nAtm(gpu_runtime::GraphT graph, 
    int32_t src, int32_t dst, int32_t edge, int32_t feat, 
    float* A, float* B, float* C, int Feat_Size) {
	// Body of the actual operator
	max_2_nAtm(dst, src, feat, A, C, Feat_Size);
}

void __device__ max_2(int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size) {
    MyatomicMax(&C[((dst * Feat_Size) + feat)], A[((src * Feat_Size) + feat)]);
}

void __device__ max_2_nAtm(int32_t src, int32_t dst, int32_t feat, float* A, float* C, int Feat_Size) {
    float tmp_d = C[((dst * Feat_Size) + feat)];
    float tmp_s = A[((src * Feat_Size) + feat)];
    C[((dst * Feat_Size) + feat)] = tmp_d >= tmp_s ? tmp_d : tmp_s;
}

void __device__ get_rst_2(int32_t v, int32_t f) {
    float tmp;
    tmp = feature_input_fc_self_2[f] + feature_input_fc_ngh_2[f];
    feature_rst_2[f] = tmp > 0 ? tmp : 0;
}

void __device__ pool_relu_2(int32_t v, int32_t f) {
    feature_input_pool_2[f] = feature_input_fc_pool_2[f] > 0 ? feature_input_fc_pool_2[f] : 0;
}

int __host__ main(int argc, char* argv[]) {
    hipError_t cuda_error;
    hipblasStatus_t cublas_error;
    // Initialize CUDA
    cuda_error = hipSetDevice(0);
    if (cuda_error != hipSuccess) {
        cout << "Error: hipSetDevice failed!" << endl;
        return EXIT_FAILURE;
    }

    // Initialize CUBLAS
    const int GPUNUM = 1;
    hipblasHandle_t* cublasHs = new hipblasHandle_t[GPUNUM];
    for (int i = 0; i < GPUNUM; i++) {
        cublas_error = hipblasCreate(&cublasHs[i]);
        if (cublas_error != HIPBLAS_STATUS_SUCCESS) {
            cout << "Error: hipblasCreate failed!" << endl;
            return EXIT_FAILURE;
        }
    }
    
	__delta_param = 1;
	gpu_runtime::load_graph(__host_edges, argv[1], false);
	hipMemcpyToSymbol(HIP_SYMBOL(edges), &__host_edges, sizeof(__host_edges), 0, hipMemcpyHostToDevice);

    // printf("Graph loaded\n");
    // printf("Graph: %d nodes, %d edges\n", __host_edges.num_vertices, __host_edges.num_edges);

	__host_edges__transposed = gpu_runtime::builtin_transpose(__host_edges);
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(edges__transposed), &__host_edges__transposed, sizeof(__host_edges__transposed), 0, hipMemcpyHostToDevice));

    // cuda_error = hipGetLastError();

    // if (cuda_error != hipSuccess) {
    //     printf("192 CUDA Error: %s\n", hipGetErrorString(cuda_error));
    // }

	__host_input_f_size = std::stoi(argv[2]);
	hipMemcpyToSymbol(HIP_SYMBOL(input_f_size), &__host_input_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_hidden_f_size = std::stoi(argv[3]);
	hipMemcpyToSymbol(HIP_SYMBOL(hidden_f_size), &__host_hidden_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_output_f_size = std::stoi(argv[4]);
	hipMemcpyToSymbol(HIP_SYMBOL(output_f_size), &__host_output_f_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
	__host_vector_size = __host_edges__transposed.num_vertices;
	hipMemcpyToSymbol(HIP_SYMBOL(vector_size), &__host_vector_size, sizeof(int32_t), 0, hipMemcpyHostToDevice);
    
    sz_fin = __host_input_f_size * __host_vector_size;
    sz_fhid = __host_hidden_f_size * __host_vector_size;
    sz_fout = __host_output_f_size * __host_vector_size;
    
    sz_w1 = __host_input_f_size * __host_hidden_f_size;
    sz_w2 = __host_hidden_f_size * __host_output_f_size;
    
    sz_w_pool1 = __host_input_f_size * __host_input_f_size;
    sz_w_pool2 = __host_hidden_f_size * __host_hidden_f_size;

    //layer-1
    hipMalloc(&__device_feature_input, sz_fin * sizeof(float));
    hipMemset(__device_feature_input, 0, sz_fin * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input), &__device_feature_input, sizeof(float*), 0);
	__host_feature_input = new float[sz_fin];
	
    hipMalloc(&__device_feature_input_pool_1, sz_fin * sizeof(float));
    hipMemset(__device_feature_input_pool_1, 0, sz_fin * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_pool_1), &__device_feature_input_pool_1, sizeof(float*), 0);
    __host_feature_input_pool_1 = new float[sz_fin];

    hipMalloc(&__device_feature_input_out_1, sz_fin * sizeof(float));
    hipMemset(__device_feature_input_out_1, 0, sz_fin * sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(feature_input_out_1), &__device_feature_input_out_1, sizeof(float*), 0);
	__host_feature_input_out_1 = new float[sz_fin];

    hipMalloc(&__device_feature_input_fc_ngh_1, sz_fhid * sizeof(float));
    hipMemset(__device_feature_input_fc_ngh_1, 0, sz_fhid * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_fc_ngh_1), &__device_feature_input_fc_ngh_1, sizeof(float*), 0);
    __host_feature_input_fc_ngh_1 = new float[sz_fhid];

    hipMalloc(&__device_feature_input_fc_self_1, sz_fhid * sizeof(float));
    hipMemset(__device_feature_input_fc_self_1, 0, sz_fhid * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_fc_self_1), &__device_feature_input_fc_self_1, sizeof(float*), 0);
    __host_feature_input_fc_self_1 = new float[sz_fhid];

    hipMalloc(&__device_feature_input_fc_pool_1, sz_fin * sizeof(float));
    hipMemset(__device_feature_input_fc_pool_1, 0, sz_fin * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_fc_pool_1), &__device_feature_input_fc_pool_1, sizeof(float*), 0);
    __host_feature_input_fc_pool_1 = new float[sz_fin];

    hipMalloc(&__device_weight_fc_ngh_1, sz_w1 * sizeof(float));
    hipMemset(__device_weight_fc_ngh_1, 0, sz_w1 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_fc_ngh_1), &__device_weight_fc_ngh_1, sizeof(float*), 0);
    __host_weight_fc_ngh_1 = new float[sz_w1];

    hipMalloc(&__device_weight_fc_self_1, sz_w1 * sizeof(float));
    hipMemset(__device_weight_fc_self_1, 0, sz_w1 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_fc_self_1), &__device_weight_fc_self_1, sizeof(float*), 0);
    __host_weight_fc_self_1 = new float[sz_w1];   

    hipMalloc(&__device_weight_pool_1, sz_w_pool1 * sizeof(float));
    hipMemset(__device_weight_pool_1, 0, sz_w_pool1 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_pool_1), &__device_weight_pool_1, sizeof(float*), 0);
    __host_weight_pool_1 = new float[sz_w_pool1];

    hipMalloc(&__device_feature_rst_1, sz_fhid * sizeof(float));
    hipMemset(__device_feature_rst_1, 0, sz_fhid * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_rst_1), &__device_feature_rst_1, sizeof(float*), 0);
    __host_feature_rst_1 = new float[sz_fhid];

    //layer-2
    hipMalloc(&__device_feature_input_pool_2, sz_fhid * sizeof(float));
    hipMemset(__device_feature_input_pool_2, 0, sz_fhid * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_pool_2), &__device_feature_input_pool_2, sizeof(float*), 0);
    __host_feature_input_pool_2 = new float[sz_fhid];

    hipMalloc(&__device_feature_input_out_2, sz_fhid * sizeof(float));
    hipMemset(__device_feature_input_out_2, 0, sz_fhid * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_out_2), &__device_feature_input_out_2, sizeof(float*), 0);
    __host_feature_input_out_2 = new float[sz_fhid];

    hipMalloc(&__device_feature_input_fc_ngh_2, sz_fout * sizeof(float));
    hipMemset(__device_feature_input_fc_ngh_2, 0, sz_fout * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_fc_ngh_2), &__device_feature_input_fc_ngh_2, sizeof(float*), 0);
    __host_feature_input_fc_ngh_2 = new float[sz_fout];

    hipMalloc(&__device_feature_input_fc_self_2, sz_fout * sizeof(float));
    hipMemset(__device_feature_input_fc_self_2, 0, sz_fout * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_fc_self_2), &__device_feature_input_fc_self_2, sizeof(float*), 0);
    __host_feature_input_fc_self_2 = new float[sz_fout];

    hipMalloc(&__device_feature_input_fc_pool_2, sz_fhid * sizeof(float));
    hipMemset(__device_feature_input_fc_pool_2, 0, sz_fhid * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_input_fc_pool_2), &__device_feature_input_fc_pool_2, sizeof(float*), 0);
    __host_feature_input_fc_pool_2 = new float[sz_fhid];

    hipMalloc(&__device_weight_fc_ngh_2, sz_w2 * sizeof(float));
    hipMemset(__device_weight_fc_ngh_2, 0, sz_w2 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_fc_ngh_2), &__device_weight_fc_ngh_2, sizeof(float*), 0);
    __host_weight_fc_ngh_2 = new float[sz_w2];

    hipMalloc(&__device_weight_fc_self_2, sz_w2 * sizeof(float));
    hipMemset(__device_weight_fc_self_2, 0, sz_w2 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_fc_self_2), &__device_weight_fc_self_2, sizeof(float*), 0);
    __host_weight_fc_self_2 = new float[sz_w2];

    hipMalloc(&__device_weight_pool_2, sz_w_pool2 * sizeof(float));
    hipMemset(__device_weight_pool_2, 0, sz_w_pool2 * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_pool_2), &__device_weight_pool_2, sizeof(float*), 0);
    __host_weight_pool_2 = new float[sz_w_pool2];

    hipMalloc(&__device_feature_rst_2, sz_fout * sizeof(float));
    hipMemset(__device_feature_rst_2, 0, sz_fout * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(feature_rst_2), &__device_feature_rst_2, sizeof(float*), 0);
    __host_feature_rst_2 = new float[sz_fout];

    // printf("init __host_weight_fc_1:%d %d %d\n", __host_input_f_size, __host_hidden_f_size, __host_hidden_f_size*__host_input_f_size);
    // initGPUData(__device_weight_fc_1, sz_w1, 0.1);
    initGPUData(__device_weight_fc_ngh_1, sz_w1, 0.001);
    initGPUData(__device_weight_fc_self_1, sz_w1, 0.001);

    // printf("init __host_weight_pool_1:%d %d %d\n", __host_input_f_size, __host_input_f_size, __host_input_f_size*__host_input_f_size);
    initGPUData(__device_weight_pool_1, sz_w_pool1, 0.001);

    initGPUData(__device_feature_input_fc_pool_1, sz_fin, 0.002);

    // printf("init __host_weight_fc_2:%d %d %d\n", __host_hidden_f_size, __host_output_f_size, __host_output_f_size*__host_hidden_f_size);
    // initGPUData(__device_weight_fc_2, sz_w2, 0.3);
    initGPUData(__device_weight_fc_ngh_2, sz_w2, 0.003);
    initGPUData(__device_weight_fc_self_2, sz_w2, 0.003);

    // printf("init __host_weight_pool_2:%d %d %d\n", __host_hidden_f_size, __host_hidden_f_size, __host_hidden_f_size*__host_hidden_f_size);
    initGPUData(__device_weight_pool_2, sz_w_pool2, 0.003);

    initGPUData(__device_feature_input_fc_pool_2, sz_fhid, 0.004);

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf("300 CUDA Error: %s\n", hipGetErrorString(cuda_error));
    }

    // decide load balance schedule
	gpu_runtime::gnn_load_balance_type lb_c[2];
    int32_t ef_num_cta[2]={0,0};
    int32_t ef_feat_size[2];
    ef_feat_size[0] = __host_input_f_size;
    ef_feat_size[1] = __host_hidden_f_size;
    int32_t group_size[2]={std::stoi(argv[9]),std::stoi(argv[13])};
    int32_t par_tiling[2]={std::stoi(argv[10]),std::stoi(argv[14])};
    gpu_runtime::uGrapher_init<gpu_operator_body_0, gpu_operator_body_0_nAtm>(
        0, argv[8], lb_c, __host_edges__transposed, ef_num_cta, 
        ef_feat_size, group_size, par_tiling);
    gpu_runtime::uGrapher_init<gpu_operator_body_1, gpu_operator_body_1_nAtm>(
        1, argv[12], lb_c, __host_edges__transposed, ef_num_cta, 
        ef_feat_size, group_size, par_tiling);
    
    gpu_runtime::f_vertex_set_apply_host<init_feature>(__host_edges__transposed, __host_input_f_size);
    
    int32_t vf_num_cta[4];
    int32_t vf_feat_size[4];
    vf_feat_size[0] = __host_input_f_size;
    vf_feat_size[1] = __host_hidden_f_size;
    vf_feat_size[2] = __host_hidden_f_size;
    vf_feat_size[3] = __host_output_f_size;
    gpu_runtime::f_vertex_set_apply_info(__host_edges__transposed, vf_num_cta[0], vf_feat_size[0]);
    gpu_runtime::f_vertex_set_apply_info(__host_edges__transposed, vf_num_cta[1], vf_feat_size[1]);
    gpu_runtime::f_vertex_set_apply_info(__host_edges__transposed, vf_num_cta[2], vf_feat_size[2]);
    gpu_runtime::f_vertex_set_apply_info(__host_edges__transposed, vf_num_cta[3], vf_feat_size[3]);

    float time = 0;
    int cnt = 20;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
	for (int32_t trail = 0; trail < cnt; trail++) {
        
        // layer-1

        // matmul fc_pool with bias
        // bias = True -> alpha = 1, beta = 1
        gpu_runtime::matmul_NN(cublasHs, __device_feature_input, __device_weight_pool_1, __device_feature_input_fc_pool_1, __host_vector_size, __host_input_f_size, __host_input_f_size, 1, 1);
        // hipDeviceSynchronize();

        {
            gpu_runtime::f_vertex_set_apply_kernel<pool_relu_1><<<vf_num_cta[0], CTA_SIZE>>>(__host_edges__transposed, vf_feat_size[0]);
            // hipDeviceSynchronize();
        }
        
        // gather
        {
            gpu_runtime::uGrapher_exec(0, lb_c, ef_num_cta, 
                __host_edges__transposed, __device_feature_input_pool_1, 0, __device_feature_input_out_1, 
                ef_feat_size, group_size, par_tiling);
            // hipDeviceSynchronize();
        }

        // matmul ngh
        // bias = False
        gpu_runtime::matmul_NN(cublasHs, __device_feature_input_out_1, __device_weight_fc_ngh_1, __device_feature_input_fc_ngh_1, __host_vector_size, __host_hidden_f_size, __host_input_f_size);

        // matmul fc_self
        gpu_runtime::matmul_NN(cublasHs, __device_feature_input, __device_weight_fc_self_1, __device_feature_input_fc_self_1, __host_vector_size, __host_hidden_f_size, __host_input_f_size);
        // hipDeviceSynchronize();

        // get_rst
        {
            gpu_runtime::f_vertex_set_apply_kernel<get_rst_1><<<vf_num_cta[1], CTA_SIZE>>>(__host_edges__transposed, vf_feat_size[1]);
            // hipDeviceSynchronize();
        }

        // layer-2
        // matmul
        gpu_runtime::matmul_NN(cublasHs, __device_feature_rst_1, __device_weight_pool_2, __device_feature_input_fc_pool_2, __host_vector_size, __host_hidden_f_size, __host_hidden_f_size, 1, 1);
        // hipDeviceSynchronize();

        {
            gpu_runtime::f_vertex_set_apply_kernel<pool_relu_2><<<vf_num_cta[2], CTA_SIZE>>>(__host_edges__transposed, vf_feat_size[2]);
            // hipDeviceSynchronize();
        }

        // gather
        {
            gpu_runtime::uGrapher_exec(1, lb_c, ef_num_cta, 
                __host_edges__transposed, __device_feature_input_pool_2, 0, __device_feature_input_out_2, 
                ef_feat_size, group_size, par_tiling);
            // hipDeviceSynchronize();
        }

        // matmul ngh
        // bias = False
        gpu_runtime::matmul_NN(cublasHs, __device_feature_input_out_2, __device_weight_fc_ngh_2, __device_feature_input_fc_ngh_2, __host_vector_size, __host_output_f_size, __host_hidden_f_size);

        // matmul fc_self
        gpu_runtime::matmul_NN(cublasHs, __device_feature_rst_1, __device_weight_fc_self_2, __device_feature_input_fc_self_2, __host_vector_size, __host_output_f_size, __host_hidden_f_size);
        // hipDeviceSynchronize();
        
        // get_rst
        {
            gpu_runtime::f_vertex_set_apply_kernel<get_rst_2><<<vf_num_cta[3], CTA_SIZE>>>(__host_edges__transposed, vf_feat_size[3]);
            // hipDeviceSynchronize();
        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    time = time / cnt;
    cout << "time: " << time << " msec" << endl;

    cuda_error = hipGetLastError();

    if (cuda_error != hipSuccess) {
        printf("Last CUDA Error: %s\n", hipGetErrorString(cuda_error));
    }

    std::fstream fp;
	fp.open(argv[15], std::ios::out|std::ios::app);
	fp << time << std::endl;
	fp.close();

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(__device_feature_input);
    hipFree(__device_feature_input_pool_1);
    hipFree(__device_feature_input_out_1);
    // hipFree(__device_feature_input_fc_1);
    hipFree(__device_feature_input_fc_pool_1);
    hipFree(__device_feature_input_fc_self_1);
    hipFree(__device_feature_input_fc_ngh_1);
    // hipFree(__device_weight_fc_1);
    hipFree(__device_weight_fc_ngh_1);
    hipFree(__device_weight_fc_self_1);
    hipFree(__device_weight_pool_1);
    hipFree(__device_feature_rst_1);
    hipFree(__device_feature_input_pool_2);
    hipFree(__device_feature_input_out_2);
    // hipFree(__device_feature_input_fc_2);
    hipFree(__device_feature_input_fc_pool_2);
    hipFree(__device_feature_input_fc_self_2);
    hipFree(__device_feature_input_fc_ngh_2);
    // hipFree(__device_weight_fc_2);
    hipFree(__device_weight_fc_ngh_2);
    hipFree(__device_weight_fc_self_2);
    hipFree(__device_weight_pool_2);
    hipFree(__device_feature_rst_2);

    delete __host_feature_input;
    delete __host_feature_input_pool_1;
    delete __host_feature_input_out_1;
    // delete __host_feature_input_fc_1;
    delete __host_feature_input_fc_pool_1;
    delete __host_feature_input_fc_self_1;
    delete __host_feature_input_fc_ngh_1;
    // delete __host_weight_fc_1;
    delete __host_weight_fc_ngh_1;
    delete __host_weight_fc_self_1;
    delete __host_weight_pool_1;
    delete __host_feature_rst_1;
    delete __host_feature_input_pool_2;
    delete __host_feature_input_out_2;
    // delete __host_feature_input_fc_2;
    delete __host_feature_input_fc_pool_2;
    delete __host_feature_input_fc_self_2;
    delete __host_feature_input_fc_ngh_2;
    // delete __host_weight_fc_2;
    delete __host_weight_fc_ngh_2;
    delete __host_weight_fc_self_2;
    delete __host_weight_pool_2;
    delete __host_feature_rst_2;

    for (int i = 0; i < GPUNUM; i++) {
		hipblasDestroy(cublasHs[i]);
	}

	delete cublasHs;
}